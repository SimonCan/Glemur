#include "hip/hip_runtime.h"
// io.cu
//
// Reading and writing routines for the snap shots and the time series.
//

#include "global.h"
#include "io.h"

// Store the information about the CUDA devices in 'cuda.info'.
int writeCudaInfo(int deviceCount, long int activeDevice)
{
    FILE                    *fd;
    struct hipDeviceProp_t   devProp;  // cuda device properties
    hipError_t             errCuda;
    int                     dev;      // device index

    fd = fopen("cuda.info", "w");
    if (fd == NULL) {
        printf("error: could not open file 'cuda.info'\n");
        exit(EXIT_FAILURE);
    }

    fprintf(fd, "number of CUDA devices: %i\n\n", deviceCount);
    fprintf(fd, "active device: %li\n\n", activeDevice);
    fprintf(fd, "PRECISION = %i\n\n", PRECISION);

    for(dev = 0; dev < deviceCount; dev++) {
        errCuda = hipGetDeviceProperties(&devProp, dev);
        if (hipSuccess != errCuda) {
            printf("error: could not obtain device properties for device number %i\n", dev);
            exit(EXIT_FAILURE); }

        fprintf(fd, "device number                   %i\n",  dev);
        fprintf(fd, "name                            %s\n",  devProp.name);
        fprintf(fd, "global memory [bytes]           %lu\n", devProp.totalGlobalMem);
        fprintf(fd, "shared memory per block [bytes] %lu\n", devProp.sharedMemPerBlock);
        fprintf(fd, "registers per block             %d\n",  devProp.regsPerBlock);
        fprintf(fd, "threads per warp                %d\n",  devProp.warpSize);
        fprintf(fd, "max memory pitch [bytes]        %lu\n", devProp.memPitch);
        fprintf(fd, "max threads per block           %d\n",  devProp.maxThreadsPerBlock);
        fprintf(fd, "max threads per block in x      %d\n",  devProp.maxThreadsDim[0]);
        fprintf(fd, "max threads per block in y      %d\n",  devProp.maxThreadsDim[1]);
        fprintf(fd, "max threads per block in z      %d\n",  devProp.maxThreadsDim[2]);
        fprintf(fd, "max blocks per grid in x        %d\n",  devProp.maxGridSize[0]);
        fprintf(fd, "max blocks per grid in y        %d\n",  devProp.maxGridSize[1]);
        fprintf(fd, "max blocks per grid in z        %d\n",  devProp.maxGridSize[2]);
        fprintf(fd, "clock rate [kHz]                %d\n",  devProp.clockRate); //
        fprintf(fd, "constant memory [bytes]         %lu\n", devProp.totalConstMem);
        fprintf(fd, "compute capability major        %d\n",  devProp.major);
        fprintf(fd, "compute capability minor        %d\n",  devProp.minor);
        fprintf(fd, "texture alignment               %lu\n", devProp.textureAlignment);
        fprintf(fd, "device overlap                  %d\n",  devProp.deviceOverlap);
        fprintf(fd, "multiprocessors                 %d\n",  devProp.multiProcessorCount);
        fprintf(fd, "kernel exec timeout             %d\n",  devProp.kernelExecTimeoutEnabled);
        fprintf(fd, "integrated GPU                  %d\n",  devProp.integrated);
        fprintf(fd, "can map host memory             %d\n",  devProp.canMapHostMemory);
        fprintf(fd, "computing mode                  %d\n",  devProp.computeMode);
        fprintf(fd, "max size 1D textures            %d\n",  devProp.maxTexture1D);
        fprintf(fd, "max size 2D textures in x       %d\n",  devProp.maxTexture2D[0]);
        fprintf(fd, "max size 2D textures in y       %d\n",  devProp.maxTexture2D[1]);
        fprintf(fd, "max size 3D textures in x       %d\n",  devProp.maxTexture3D[0]);
        fprintf(fd, "max size 3D textures in y       %d\n",  devProp.maxTexture3D[1]);
        fprintf(fd, "max size 3D textures in z       %d\n",  devProp.maxTexture3D[2]);
        fprintf(fd, "concurrent kernels              %d\n",  devProp.concurrentKernels);
        fprintf(fd, "ECC enabled                     %d\n",  devProp.ECCEnabled);
        fprintf(fd, "PCI bus ID                      %d\n",  devProp.pciBusID);
        fprintf(fd, "PCI device ID                   %d\n",  devProp.pciDeviceID);
        fprintf(fd, "PCI domain ID                   %d\n",  devProp.pciDomainID);
        fprintf(fd, "TCC drive                       %d\n",  devProp.tccDriver);
        fprintf(fd, "asynchronous engine count       %d\n",  devProp.asyncEngineCount);
        fprintf(fd, "unified addressing with host    %d\n",  devProp.unifiedAddressing);
        fprintf(fd, "memory clock rate [kHz]         %d\n",  devProp.memoryClockRate);
        fprintf(fd, "memory bus width [bits]         %d\n",  devProp.memoryBusWidth);
        fprintf(fd, "l2 cache size [bytes]           %d\n",  devProp.l2CacheSize);
        fprintf(fd, "maximum threads per multiproc.  %d\n",  devProp.maxThreadsPerMultiProcessor);
        fprintf(fd, "\n");
    }

    fclose(fd);

    return 0;
}


// Swap byte order for the correct endianess.
REAL floatSwap(REAL value) {
    union v {
        REAL  f;
        UINT  i;
    };

    union  v val;
    #if (PRECISION == 64)
        unsigned int left, right;
    #endif

    if (endian == LITTLE_ENDIAN) {
        val.f = value;
        #if (PRECISION == 32)
            val.i  = htonl(val.i);
        #endif
        #if (PRECISION == 64)
            right = (unsigned int) val.i;
            val.i = val.i >> 32;
            left  = (unsigned int) val.i;
            left  = htonl(left);
            right = htonl(right);
            val.i = right;
            val.i = val.i << 32;
            val.i = val.i | (UINT) left;
        #endif
        return *(REAL*)&val.i;
    }
    else
        return value;
}


// Swap byte order for the correct endianess.
int intSwap(int value){
    union v {
        int             f;
        unsigned int    i;
    };

    union        v val;
    unsigned int temp;

    val.f = value;
    temp  = htonl(val.i);

    if (endian == LITTLE_ENDIAN)
        return *(int*)&temp;
    else
        return value;
}


// Eead the number of grid points from the last snapshot.
int readGrid(struct Parameters params)
{
    FILE *  fd;
    char    line[256];   // text line in the save file
    char    tmp[256];
    int     i;

    fd = fopen("data/save.vtk", "r");
    if (fd == NULL) {
        printf("error: could not open file 'data/save.vtk'\n");
        exit(EXIT_FAILURE);
    }

    // Jump over the header.
    for (i = 0; i < 5; i++)
        fgets(line, sizeof(line), fd);

    strncpy(tmp, line+11, 9); params.nx = atoi(tmp)-2;
    strncpy(tmp, line+21, 9); params.ny = atoi(tmp)-2;
    strncpy(tmp, line+31, 9); params.nz = atoi(tmp)-2;

    fclose(fd);
    return 0;
}


// Dump the initial magnetic field.
int writeB0(REAL *B0, struct Parameters params)
{
    FILE    *fd;
    int     i, j, k;
    REAL    swapped[1]; // needed to swap byte order to big endian

    fd = fopen("data/B0.vtk", "w");
    if (fd == NULL) {
        printf("error: could not open file 'data/B0.vtk'\n");
        exit(EXIT_FAILURE);
    }

    // Write common header.
    fprintf(fd, "# vtk DataFile Version 2.0\n");
    fprintf(fd, "GLEMuR B0 dump\n");
    fprintf(fd, "BINARY\n");
    fprintf(fd, "DATASET STRUCTURED_POINTS\n");
    fprintf(fd, "DIMENSIONS %9i %9i %9i\n", params.nx+2, params.ny+2, params.nz+2);
    fprintf(fd, "ORIGIN %8.12f %8.12f %8.12f\n", params.Ox-params.dx, params.Oy-params.dy, params.Oz-params.dz);
    fprintf(fd, "SPACING %8.12f %8.12f %8.12f\n", params.Lx/(params.nx-1), params.Ly/(params.ny-1), params.Lz/(params.nz-1));
    fprintf(fd, "POINT_DATA %9i\n", (params.nx+2)*(params.ny+2)*(params.nz+2));
    fprintf(fd, "VECTORS bfield %s\n", REAL_STR);

    for (k = 0; k < params.nz+2; k++) {
        for (j = 0; j < params.ny+2; j++) {
            for (i = 0; i < params.nx+2; i++) {
                swapped[0] = floatSwap(B0[0 + i*3 + j*(params.nx+2)*3 + k*(params.nx+2)*(params.ny+2)*3]);
                fwrite(swapped, sizeof(REAL), 1, fd);
                swapped[0] = floatSwap(B0[1 + i*3 + j*(params.nx+2)*3 + k*(params.nx+2)*(params.ny+2)*3]);
                fwrite(swapped, sizeof(REAL), 1, fd);
                swapped[0] = floatSwap(B0[2 + i*3 + j*(params.nx+2)*3 + k*(params.nx+2)*(params.ny+2)*3]);
                fwrite(swapped, sizeof(REAL), 1, fd);
            }
        }
    }

    fclose(fd);
    return 0;
}


// Dump the current state.
int dumpState(struct VarsHost h, struct Parameters params, REAL t, REAL dt, int n)
{
    FILE    *fd;
    int     i, j, k;
    REAL    swapped[1];     // needed to swap byte order to big endian
    char    fileName[20];

    if (n == -1)
        sprintf(fileName, "data/save.vtk");
    else
        sprintf(fileName, "data/dump%d.vtk", n);
    fd = fopen(fileName, "w");

    if (fd == NULL) {
        printf("error: could not open file '%s'\n", fileName);
        exit(EXIT_FAILURE);
    }

    // Write common header.
    fprintf(fd, "# vtk DataFile Version 2.0\n");
    fprintf(fd, "GLEMuR data dump\n");
    fprintf(fd, "BINARY\n");
    fprintf(fd, "DATASET STRUCTURED_GRID\n");
    fprintf(fd, "DIMENSIONS %9i %9i %9i\n", params.nx+2, params.ny+2, params.nz+2);

    // Parameters as meta data.
    writeParams(params, t, dt, fd);

    // Write structured grid xb.
    fprintf(fd, "POINTS %9i %s\n", (params.nx+2)*(params.ny+2)*(params.nz+2), REAL_STR);
    for (k = 0; k < params.nz+2; k++) {
        for (j = 0; j < params.ny+2; j++) {
            for (i = 0; i < params.nx+2; i++) {
                swapped[0] = floatSwap(h.xb[0 + i*3 + j*(params.nx+2)*3 + k*(params.nx+2)*(params.ny+2)*3]);
                fwrite(swapped, sizeof(REAL), 1, fd);
                swapped[0] = floatSwap(h.xb[1 + i*3 + j*(params.nx+2)*3 + k*(params.nx+2)*(params.ny+2)*3]);
                fwrite(swapped, sizeof(REAL), 1, fd);
                swapped[0] = floatSwap(h.xb[2 + i*3 + j*(params.nx+2)*3 + k*(params.nx+2)*(params.ny+2)*3]);
                fwrite(swapped, sizeof(REAL), 1, fd);
            }
        }
    }

    fprintf(fd, "POINT_DATA %9i\n", (params.nx+2)*(params.ny+2)*(params.nz+2));

    // Write magnetic field B.
    fprintf(fd, "VECTORS bfield %s\n", REAL_STR);
    for (k = 0; k < params.nz+2; k++) {
        for (j = 0; j < params.ny+2; j++) {
            for (i = 0; i < params.nx+2; i++) {
                swapped[0] = floatSwap(h.B[0 + i*3 + j*(params.nx+2)*3 + k*(params.nx+2)*(params.ny+2)*3]);
                fwrite(swapped, sizeof(REAL), 1, fd);
                swapped[0] = floatSwap(h.B[1 + i*3 + j*(params.nx+2)*3 + k*(params.nx+2)*(params.ny+2)*3]);
                fwrite(swapped, sizeof(REAL), 1, fd);
                swapped[0] = floatSwap(h.B[2 + i*3 + j*(params.nx+2)*3 + k*(params.nx+2)*(params.ny+2)*3]);
                fwrite(swapped, sizeof(REAL), 1, fd);
            }
        }
    }

    // Write velocity field uu.
    if (params.inertia == 1) {
        fprintf(fd, "VECTORS ufield %s\n", REAL_STR);
        for (k = 0; k < params.nz+2; k++) {
            for (j = 0; j < params.ny+2; j++) {
                for (i = 0; i < params.nx+2; i++) {
                    if ((i == 0) || (i == params.nx+1) || (j == 0) || (j == params.ny+1) || (k == 0) || (k == params.nz+1)) {
                        swapped[0] = 0.;
                        fwrite(swapped, sizeof(REAL), 1, fd);
                        fwrite(swapped, sizeof(REAL), 1, fd);
                        fwrite(swapped, sizeof(REAL), 1, fd);
                    }
                    else {
                        swapped[0] = floatSwap(h.uu[0 + (i-1)*3 + (j-1)*params.nx*3 + (k-1)*params.nx*params.ny*3]);
                        fwrite(swapped, sizeof(REAL), 1, fd);
                        swapped[0] = floatSwap(h.uu[1 + (i-1)*3 + (j-1)*params.nx*3 + (k-1)*params.nx*params.ny*3]);
                        fwrite(swapped, sizeof(REAL), 1, fd);
                        swapped[0] = floatSwap(h.uu[2 + (i-1)*3 + (j-1)*params.nx*3 + (k-1)*params.nx*params.ny*3]);
                        fwrite(swapped, sizeof(REAL), 1, fd);
                    }
                }
            }
        }
    }

    // Write electric current density J.
    if (params.dumpJ == 1) {
        fprintf(fd, "VECTORS jfield %s\n", REAL_STR);
        for (k = 0; k < params.nz+2; k++) {
            for (j = 0; j < params.ny+2; j++) {
                for (i = 0; i < params.nx+2; i++) {
                    if ((i == 0) || (i == params.nx+1) || (j == 0) || (j == params.ny+1) || (k == 0) || (k == params.nz+1)) {
                        swapped[0] = 0.;
                        fwrite(swapped, sizeof(REAL), 1, fd);
                        fwrite(swapped, sizeof(REAL), 1, fd);
                        fwrite(swapped, sizeof(REAL), 1, fd);
                    }
                    else {
                        swapped[0] = floatSwap(h.JJ[0 + (i-1)*3 + (j-1)*params.nx*3 + (k-1)*params.nx*params.ny*3]);
                        fwrite(swapped, sizeof(REAL), 1, fd);
                        swapped[0] = floatSwap(h.JJ[1 + (i-1)*3 + (j-1)*params.nx*3 + (k-1)*params.nx*params.ny*3]);
                        fwrite(swapped, sizeof(REAL), 1, fd);
                        swapped[0] = floatSwap(h.JJ[2 + (i-1)*3 + (j-1)*params.nx*3 + (k-1)*params.nx*params.ny*3]);
                        fwrite(swapped, sizeof(REAL), 1, fd);
                    }
                }
            }
        }
    }

    // Write the determinant of the Jacobian matrix.
    if (params.dumpDetJac == 1) {
        fprintf(fd, "SCALARS detJac %s\n", REAL_STR);
        fprintf(fd, "LOOKUP_TABLE default\n");
        for (k = 0; k < params.nz+2; k++) {
            for (j = 0; j < params.ny+2; j++) {
                for (i = 0; i < params.nx+2; i++) {
                    swapped[0] = floatSwap(h.detJac[i + j*(params.nx+2) + k*(params.nx+2)*(params.ny+2)]);
                    fwrite(swapped, sizeof(REAL), 1, fd);
                }
            }
        }
    }

    // Write the signed cell volume.
    if (params.dumpCellVol == 1) {
        fprintf(fd, "SCALARS cellVol %s\n", REAL_STR);
        fprintf(fd, "LOOKUP_TABLE default\n");
        for (k = 0; k < params.nz+2; k++) {
            for (j = 0; j < params.ny+2; j++) {
                for (i = 0; i < params.nx+2; i++) {
                    if ((i*j*k == 0) || (i == params.nx+1) || (j == params.ny+1) || (k == params.nz+1))
                        swapped[0] = floatSwap(0);
                    else
                        swapped[0] = floatSwap(h.cellVol[(i-1) + (j-1)*params.nx + (k-1)*params.nx*params.ny]);
                    fwrite(swapped, sizeof(REAL), 1, fd);
                }
            }
        }
    }

    // Write the convexity of the cells around the grid point.
    if (params.dumpConvexity == 1) {
        fprintf(fd, "SCALARS convexity %s\n", REAL_STR);
        fprintf(fd, "LOOKUP_TABLE default\n");
        for (k = 0; k < params.nz+2; k++) {
            for (j = 0; j < params.ny+2; j++) {
                for (i = 0; i < params.nx+2; i++) {
                    if ((i*j*k == 0) || (i == params.nx+1) || (j == params.ny+1) || (k == params.nz+1))
                        swapped[0] = floatSwap(0);
                    else
                        swapped[0] = floatSwap(h.convexity[(i-1) + (j-1)*params.nx + (k-1)*params.nx*params.ny]);
                    fwrite(swapped, sizeof(REAL), 1, fd);
                }
            }
        }
    }

    // Write the minimum of the wedge products.
    if (params.dumpWedgeMin == 1) {
        fprintf(fd, "SCALARS wedgeMin %s\n", REAL_STR);
        fprintf(fd, "LOOKUP_TABLE default\n");
        for (k = 0; k < params.nz+2; k++) {
            for (j = 0; j < params.ny+2; j++) {
                for (i = 0; i < params.nx+2; i++) {
                    if ((i*j*k == 0) || (i == params.nx+1) || (j == params.ny+1) || (k == params.nz+1))
                        swapped[0] = floatSwap(0);
                    else
                        swapped[0] = floatSwap(h.wedgeMin[(i-1) + (j-1)*params.nx + (k-1)*params.nx*params.ny]);
                    fwrite(swapped, sizeof(REAL), 1, fd);
                }
            }
        }
    }

    fclose(fd);
    return 0;
}


// Read the state and the parameters from the latest dump file.
int readState(struct VarsHost h, struct Parameters params, REAL *t, REAL *dt)
{
    FILE *  fd;
    char    line[256];   // text line in the save file
    char    tmp[256];
    char    *pos;        // position in a string
    int     numParams;   // number of fields with parameters
    int     i, j, k, l;
    REAL    legacy;      // for backwards compatibility with older files

    fd = fopen("data/save.vtk", "r");
    if (fd == NULL) {
        printf("error: could not open file 'data/save.vtk'\n");
        exit(EXIT_FAILURE);
    }

    // Jump over the header.
    for (i = 0; i < 5; i++)
        fgets(line, sizeof(line), fd);

    // Read how many fields there are.
    fgets(line, sizeof(line), fd); strncpy(tmp, line+17, 2);
    numParams = atoi(tmp);

    // Write the parameters into the parameters struct.
    for (i = 0; i < numParams; i++) {
        fgets(line, sizeof(line), fd);
        pos = strstr(line, " ");
        strncpy(tmp, line, pos-line);

        if (strncmp(tmp, "t", pos-line) == 0) {
            fread(t, sizeof(REAL), 1, fd); *t = floatSwap(*t);
        }
        if (strncmp(tmp, "dt", pos-line) == 0) {
            fread(dt, sizeof(REAL), 1, fd); *dt = floatSwap(*dt);
        }
        if (strncmp(tmp, "nx_ny_nz", pos-line) == 0) {
            fread(&params.nx, sizeof(int), 1, fd); params.nx = intSwap(params.nx);
            fread(&params.ny, sizeof(int), 1, fd); params.ny = intSwap(params.ny);
            fread(&params.nz, sizeof(int), 1, fd); params.nz = intSwap(params.nz);
        }
        if (strncmp(tmp, "Lx_Ly_Lz", pos-line) == 0) {
            fread(&params.Lx, sizeof(REAL), 1, fd); params.Lx = floatSwap(params.Lx);
            fread(&params.Ly, sizeof(REAL), 1, fd); params.Ly = floatSwap(params.Ly);
            fread(&params.Lz, sizeof(REAL), 1, fd); params.Lz = floatSwap(params.Lz);
        }
        if (strncmp(tmp, "Ox_Oy_Oz", pos-line) == 0) {
            fread(&params.Ox, sizeof(REAL), 1, fd); params.Ox = floatSwap(params.Ox);
            fread(&params.Oy, sizeof(REAL), 1, fd); params.Oy = floatSwap(params.Oy);
            fread(&params.Oz, sizeof(REAL), 1, fd); params.Oz = floatSwap(params.Oz);
        }
        if (strncmp(tmp, "dx_dy_dz", pos-line) == 0) {
            fread(&params.dx, sizeof(REAL), 1, fd); params.dx = floatSwap(params.dx);
            fread(&params.dy, sizeof(REAL), 1, fd); params.dy = floatSwap(params.dy);
            fread(&params.dz, sizeof(REAL), 1, fd); params.dz = floatSwap(params.dz);
        }
        if (strncmp(tmp, "ampl", pos-line) == 0) {
            fread(&params.ampl, sizeof(REAL), 1, fd); params.ampl = floatSwap(params.ampl);
        }
        if (strncmp(tmp, "phi1_phi2", pos-line) == 0) {
            fread(&params.phi1, sizeof(REAL), 1, fd); params.phi1 = floatSwap(params.phi1);
            fread(&params.phi2, sizeof(REAL), 1, fd); params.phi2 = floatSwap(params.phi2);
        }
        if (strncmp(tmp, "rxhalf_ryhalf", pos-line) == 0) {
            fread(&legacy, sizeof(REAL), 1, fd);
            fread(&legacy, sizeof(REAL), 1, fd);
        }
    }

    // Read the grid data.
    fgets(line, sizeof(line), fd);
    fread(h.xb, sizeof(REAL), 3*(params.nx+2)*(params.ny+2)*(params.nz+2), fd);
    for (i = 0; i < 3*(params.nx+2)*(params.ny+2)*(params.nz+2); i++)
        h.xb[i] = floatSwap(h.xb[i]);

    // Read the velocity field. need to jump over B-field first.
    if (params.inertia == true) {
        REAL* uu_tmp;
        uu_tmp = (REAL *)malloc(3*(params.nx+2)*(params.ny+2)*(params.nz+2)*sizeof(*(uu_tmp)));
        if (uu_tmp == NULL) { printf("error: could not allocate memory for uu_tmp\n"); return -1; }
        fgets(line, sizeof(line), fd);
        fread(uu_tmp, sizeof(REAL), 3*(params.nx+2)*(params.ny+2)*(params.nz+2), fd);    // jump over B
        fgets(line, sizeof(line), fd);
        fread(uu_tmp, sizeof(REAL), 3*(params.nx+2)*(params.ny+2)*(params.nz+2), fd);
        for (k = 0; k < params.nz; k++)
            for (j = 0; j < params.ny; j++)
                for (i = 0; i < params.nx; i++)
                    for (l = 0; l < 3; l++)
                        h.uu[l + 3*i + 3*params.nx*j + 3*params.nx*params.ny*k] = floatSwap(uu_tmp[l + 3*(i+1) + 3*(params.nx+2)*(j+1) + 3*(params.nx+2)*(params.ny+2)*(k+1)]);
    }

    fclose(fd);

    // Read the initial magnetic field B0.
    fd = fopen("data/B0.vtk", "r");
    if (fd == NULL) {
        printf("error: could not open file 'data/B0.vtk'\n");
        exit(EXIT_FAILURE);
    }
    for (i = 0; i < 9; i++)
        fgets(line, sizeof(line), fd);

    fread(h.B0, sizeof(REAL), 3*(params.nx+2)*(params.ny+2)*(params.nz+2), fd);
    for (i = 0; i < 3*(params.nx+2)*(params.ny+2)*(params.nz+2); i++)
        h.B0[i] = floatSwap(h.B0[i]);

    fclose(fd);

    return 0;
}


// Write parameters in the dumping files.
int writeParams(struct Parameters params, REAL t, REAL dt, FILE *fd)
{
    REAL    swappedF[1]; // needed to swap byte order to big endian
    int     swappedI[1]; // needed to swap byte order to big endian

    fprintf(fd, "FIELD parameters %i\n", 9);
    fprintf(fd, "t 1 1 %s\n", REAL_STR);
    swappedF[0] = floatSwap(t); fwrite(swappedF, sizeof(REAL), 1, fd);
    fprintf(fd, "dt 1 1 %s\n", REAL_STR);
    swappedF[0] = floatSwap(dt); fwrite(swappedF, sizeof(REAL), 1, fd);
    fprintf(fd, "nx_ny_nz 1 3 int\n");
    swappedI[0] = intSwap(params.nx); fwrite(swappedI, sizeof(int), 1, fd);
    swappedI[0] = intSwap(params.ny); fwrite(swappedI, sizeof(int), 1, fd);
    swappedI[0] = intSwap(params.nz); fwrite(swappedI, sizeof(int), 1, fd);
    fprintf(fd, "Lx_Ly_Lz 1 3 %s\n", REAL_STR);
    swappedF[0] = floatSwap(params.Lx); fwrite(swappedF, sizeof(REAL), 1, fd);
    swappedF[0] = floatSwap(params.Ly); fwrite(swappedF, sizeof(REAL), 1, fd);
    swappedF[0] = floatSwap(params.Lz); fwrite(swappedF, sizeof(REAL), 1, fd);
    fprintf(fd, "Ox_Oy_Oz 1 3 %s\n", REAL_STR);
    swappedF[0] = floatSwap(params.Ox); fwrite(swappedF, sizeof(REAL), 1, fd);
    swappedF[0] = floatSwap(params.Oy); fwrite(swappedF, sizeof(REAL), 1, fd);
    swappedF[0] = floatSwap(params.Oz); fwrite(swappedF, sizeof(REAL), 1, fd);
    fprintf(fd, "dx_dy_dz 1 3 %s\n", REAL_STR);
    swappedF[0] = floatSwap(params.dx); fwrite(swappedF, sizeof(REAL), 1, fd);
    swappedF[0] = floatSwap(params.dy); fwrite(swappedF, sizeof(REAL), 1, fd);
    swappedF[0] = floatSwap(params.dz); fwrite(swappedF, sizeof(REAL), 1, fd);
    fprintf(fd, "ampl 1 1 %s\n", REAL_STR);
    swappedF[0] = floatSwap(params.ampl); fwrite(swappedF, sizeof(REAL), 1, fd);
    fprintf(fd, "phi1_phi2 1 2 %s\n", REAL_STR);
    swappedF[0] = floatSwap(params.phi1); fwrite(swappedF, sizeof(REAL), 1, fd);
    swappedF[0] = floatSwap(params.phi2); fwrite(swappedF, sizeof(REAL), 1, fd);
    fprintf(fd, "maxError 1 1 %s\n", REAL_STR);
    swappedF[0] = floatSwap(params.maxError); fwrite(swappedF, sizeof(REAL), 1, fd);

    return 0;
}


// Write out the time series.
int writeTs(struct Parameters params, struct Reduction red, REAL t, REAL dt, int it, REAL maxDelta)
{
    FILE  * fd;

    if (t == 0) {
        fd = fopen("data/time_series.dat", "w+");
        if (fd == NULL) {
            printf("error: could not open file 'data/time_series.dat'\n");
            exit(EXIT_FAILURE);
        }

        printf("#%7s", "it");
        printf("%13s", "t");
        printf("%13s", "dt");
        printf("%13s", "maxError");
        if (params.redJMax == true)
            printf("%13s", "JMax");
        if (params.redJxB_B2Max == true)
            printf("%13s", "JxB_B2Max");
        if (params.redEpsilonStar == true)
            printf("%13s", "epsilonStar");
        if (params.redErrB_1ez == true)
            printf("%13s", "errB_1ez");
        if (params.redErrXb_XbAn == true)
            printf("%13s", "errXb_XbAn");
        if (params.redB2 == true)
            printf("%13s", "B2");
        if (params.redB2f == true)
            printf("%13s", "B2f");
        if (params.redConvex == true)
            printf("%13s", "convex");
        if (params.redWedgeMin == true)
            printf("%13s", "wedgeMin");
        if ((params.redU2 == true) && (params.inertia == true))
            printf("%13s", "U2");
        if ((params.redUMax == true) && (params.inertia == true))
            printf("%13s", "UMax");
        printf("\n");

        fprintf(fd, "#%7s", "it");
        fprintf(fd, "%13s", "t");
        fprintf(fd, "%13s", "dt");
        fprintf(fd, "%13s", "maxDelta");
        if (params.redJMax == true)
            fprintf(fd, "%13s", "JMax");
        if (params.redJxB_B2Max == true)
            fprintf(fd, "%13s", "JxB_B2Max");
        if (params.redEpsilonStar == true)
            fprintf(fd, "%13s", "epsilonStar");
        if (params.redErrB_1ez == true)
            fprintf(fd, "%13s", "errB_1ez");
        if (params.redErrXb_XbAn == true)
            fprintf(fd, "%13s", "errXb_XbAn");
        if (params.redB2 == true)
            fprintf(fd, "%13s", "B2");
        if (params.redB2f == true)
            fprintf(fd, "%13s", "B2f");
        if (params.redConvex == true)
            fprintf(fd, "%13s", "convex");
        if (params.redWedgeMin == true)
            fprintf(fd, "%13s", "wedgeMin");
        if ((params.redU2 == true) && (params.inertia == true))
            fprintf(fd, "%13s", "U2");
        if ((params.redUMax == true) && (params.inertia == true))
            fprintf(fd, "%13s", "UMax");
        fprintf(fd, "\n");

        fclose(fd);
    }
    else {
        fd = fopen("data/time_series.dat", "a");
        if (fd == NULL) {
            printf("error: could not open file 'data/time_series.dat'\n");
            exit(EXIT_FAILURE);
        }

        printf("%8i ", it);
        printf("%12.5e ", t);
        printf("%12.5e ", dt);
        printf("%12.5e ", maxDelta);
        if (params.redJMax == true)
            printf("%12.5e ", red.JMax);
        if (params.redJxB_B2Max == true)
            printf("%12.5e ", red.JxB_B2Max);
        if (params.redEpsilonStar == true)
            printf("%12.5e ", red.epsilonStar);
        if (params.redErrB_1ez == true)
            printf("%12.5e ", red.errB_1ez);
        if (params.redErrXb_XbAn == true)
            printf("%12.5e ", red.errXb_XbAn);
        if (params.redB2 == true)
            printf("%12.5e ", red.B2);
        if (params.redB2f == true)
            printf("%12.5e ", red.B2-red.B2res);
        if (params.redConvex == true)
            printf("%12.5e ", red.convex);
        if (params.redWedgeMin == true)
            printf("%12.5e ", red.wedgeMin);
        if ((params.redU2 == true) && (params.inertia == true))
            printf("%12.5e ", red.U2);
        if ((params.redUMax == true) && (params.inertia == true))
            printf("%12.5e ", red.UMax);
        printf("\n");

        fprintf(fd, "%8i ", it);
        fprintf(fd, "%12.5e ", t);
        fprintf(fd, "%12.5e ", dt);
        fprintf(fd, "%12.5e ", maxDelta);
        if (params.redJMax == true)
            fprintf(fd, "%12.5e ", red.JMax);
        if (params.redJxB_B2Max == true)
            fprintf(fd, "%12.5e ", red.JxB_B2Max);
        if (params.redEpsilonStar == true)
            fprintf(fd, "%12.5e ", red.epsilonStar);
        if (params.redErrB_1ez == true)
            fprintf(fd, "%12.5e ", red.errB_1ez);
        if (params.redErrXb_XbAn == true)
            fprintf(fd, "%12.5e ", red.errXb_XbAn);
        if (params.redB2 == true)
            fprintf(fd, "%12.5e ", red.B2);
        if (params.redB2f == true)
            fprintf(fd, "%12.5e ", red.B2-red.B2res);
        if (params.redConvex == true)
            fprintf(fd, "%12.5e ", red.convex);
        if (params.redWedgeMin == true)
            fprintf(fd, "%12.5e ", red.wedgeMin);
        if ((params.redU2 == true) && (params.inertia == true))
            fprintf(fd, "%12.5e ", red.U2);
        if ((params.redUMax == true) && (params.inertia == true))
            fprintf(fd, "%12.5e ", red.UMax);
        fprintf(fd, "\n");

        fclose(fd);
    }

    return 0;
}
