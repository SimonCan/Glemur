// memory.cu
//
// Memory allocation and freeing routines for host and device memory.
//

#include "memory.h"
#include "global.h"

// allocate host and device memory
int allocateMemory(struct varsHost_t *h, struct varsDev_t *d, struct parameters_t p, int gridSize[3]) {
	hipError_t         errCuda;      	// error returned by device functions

	// allocate host memory
    h->B0 = (REAL *)malloc(3*(p.nx+2)*(p.ny+2)*(p.nz+2)*sizeof(*(h->B0)));
    if (h->B0 == NULL) { printf("error: could not allocate memory for B0\n"); return -1; }
    h->B = (REAL *)malloc(3*(p.nx+2)*(p.ny+2)*(p.nz+2)*sizeof(*(h->B)));
    if (h->B == NULL) { printf("error: could not allocate memory for B\n"); return -1; }
    h->xb = (REAL *)malloc(3*(p.nx+2)*(p.ny+2)*(p.nz+2)*sizeof(*(h->xb)));
    if (h->xb == NULL) { printf("error: could not allocate memory for xb\n"); return -1; }
    h->JJ = (REAL *)malloc(3*p.nx*p.ny*p.nz*sizeof(*(h->JJ)));
    if (h->JJ == NULL) { printf("error: could not allocate memory for JJ\n"); return -1; }
    h->detJac = (REAL *)malloc((p.nx+2)*(p.ny+2)*(p.nz+2)*sizeof(*(h->detJac)));
    if (h->detJac == NULL) { printf("error: could not allocate memory for detJac\n"); return -1; }
    if (p.dumpCellVol == 1) {
    	h->cellVol = (REAL *)malloc(p.nx*p.ny*p.nz*sizeof(*(h->cellVol)));
        if (h->cellVol == NULL) { printf("error: could not allocate memory for cellVol\n"); return -1; } }
    if ((p.dumpConvexity == 1) || (p.redConvex == 1)) {
    	h->convexity = (REAL *)malloc(p.nx*p.ny*p.nz*sizeof(*(h->convexity)));
        if (h->cellVol == NULL) { printf("error: could not allocate memory for convexity\n"); return -1; } }
    if ((p.dumpWedgeMin == 1) || (p.redWedgeMin == 1)) {
    	h->wedgeMin = (REAL *)malloc(p.nx*p.ny*p.nz*sizeof(*(h->wedgeMin)));
        if (h->wedgeMin == NULL) { printf("error: could not allocate memory for wedgeMin\n"); return -1; } }
    if (p.inertia == true) {
    	h->uu = (REAL *)malloc(3*p.nx*p.ny*p.nz*sizeof(*(h->uu)));
        if (h->uu == NULL) { printf("error: could not allocate memory for uu\n"); return -1; } }

    // allocate global memory on GPU device
    errCuda = hipMalloc((void**)&(d->B0), 3*(p.nx+2)*(p.ny+2)*(p.nz+2)*sizeof(*(d->B0)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for B0\n"); exit(EXIT_FAILURE); }
    errCuda = hipMalloc((void**)&(d->B), 3*(p.nx+2)*(p.ny+2)*(p.nz+2)*sizeof(*(d->B)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for B\n"); exit(EXIT_FAILURE); }
    errCuda = hipMalloc((void**)&(d->xb), 3*(p.nx+2)*(p.ny+2)*(p.nz+2)*sizeof(*(d->xb)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for xb\n"); exit(EXIT_FAILURE); }
    errCuda = hipMalloc((void**)&(d->J), 3*p.nx*p.ny*p.nz*sizeof(*(d->J)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for J\n"); exit(EXIT_FAILURE); }
    errCuda = hipMalloc((void**)&(d->xb_new), 3*(p.nx+2)*(p.ny+2)*(p.nz+2)*sizeof(*(d->xb_new)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for xb_new\n"); exit(EXIT_FAILURE); }
    errCuda = hipMalloc((void**)&(d->xb_tmp), 3*(p.nx+2)*(p.ny+2)*(p.nz+2)*sizeof(*(d->xb_tmp)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for xb_tmp\n"); exit(EXIT_FAILURE); }
    if (p.inertia == false) {
		errCuda = hipMalloc((void**)&(d->kk), 3*p.nx*p.ny*p.nz*6*sizeof(*(d->kk)));
		if (hipSuccess != errCuda) { printf("error: could not allocate device memory for kk\n"); exit(EXIT_FAILURE); } }
	else {
		errCuda = hipMalloc((void**)&(d->kk), 6*p.nx*p.ny*p.nz*6*sizeof(*(d->kk)));
		if (hipSuccess != errCuda) { printf("error: could not allocate device memory for kk\n"); exit(EXIT_FAILURE); } }
    if (p.pressure == true) {
        errCuda = hipMalloc((void**)&(d->gradP), 3*p.nx*p.ny*p.nz*sizeof(*(d->gradP)));
        if (hipSuccess != errCuda) { printf("error: could not allocate device memory for gradP\n"); exit(EXIT_FAILURE); } }
    errCuda = hipMalloc((void**)&(d->maxDelta), gridSize[0]*gridSize[1]*gridSize[2]*sizeof(*(d->maxDelta)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for maxDelta\n"); exit(EXIT_FAILURE); }
    errCuda = hipMalloc((void**)&(d->detJac), (p.nx+2)*(p.ny+2)*(p.nz+2)*sizeof(*(d->detJac)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for detJac\n"); exit(EXIT_FAILURE); }
    if (p.inertia == true) {
        errCuda = hipMalloc((void**)&(d->uu), 3*p.nx*p.ny*p.nz*sizeof(*(d->uu)));
        if (hipSuccess != errCuda) { printf("error: could not allocate device memory for uu\n"); exit(EXIT_FAILURE); }
        errCuda = hipMalloc((void**)&(d->uu_new), 3*p.nx*p.ny*p.nz*sizeof(*(d->uu_new)));
        if (hipSuccess != errCuda) { printf("error: could not allocate device memory for uu_new\n"); exit(EXIT_FAILURE); }
        errCuda = hipMalloc((void**)&(d->uu_tmp), 3*p.nx*p.ny*p.nz*sizeof(*(d->uu_tmp)));
        if (hipSuccess != errCuda) { printf("error: could not allocate device memory for uu_tmp\n"); exit(EXIT_FAILURE); }
    }

    return 0;
}


// free host and device memory
int freeMemory(struct varsHost_t *h, struct varsDev_t *d, struct parameters_t p) {
    free(h->B0); free(h->B); free(h->JJ); free(h->xb); free(h->detJac);
    hipFree(d->kk); hipFree(d->xb_new); hipFree(d->maxDelta); hipFree(d->xb_tmp);
    hipFree(d->detJac);
    hipFree(d->xb); hipFree(d->B0); hipFree(d->B); hipFree(d->J);
    if (p.dumpCellVol == 1)
    	free(h->cellVol);
    if ((p.dumpConvexity == 1) || (p.redConvex == 1))
    	free(h->convexity);
    if ((p.dumpWedgeMin == 1) || (p.redWedgeMin == 1))
    	free(h->wedgeMin);
    if (p.inertia == true) {
    	free(h->uu);
    	hipFree(d->uu); hipFree(d->uu_new); hipFree(d->uu_tmp);
    }

    return 0;
}

