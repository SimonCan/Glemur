// memory.cu
//
// Memory allocation and freeing routines for host and device memory.
//

#include "memory.h"
#include "global.h"

// Allocate host and device memory.
int allocateMemory(struct VarsHost *h, struct VarsDev *d, struct Parameters params, int gridSize[3]) {
    hipError_t	errCuda;	// error returned by device functions

    // Allocate host memory.
    h->B0 = (REAL *)malloc(3*(params.nx+2)*(params.ny+2)*(params.nz+2)*sizeof(*(h->B0)));
    if (h->B0 == NULL) { printf("error: could not allocate memory for B0\n"); return -1; }
    h->B = (REAL *)malloc(3*(params.nx+2)*(params.ny+2)*(params.nz+2)*sizeof(*(h->B)));
    if (h->B == NULL) { printf("error: could not allocate memory for B\n"); return -1; }
    h->xb = (REAL *)malloc(3*(params.nx+2)*(params.ny+2)*(params.nz+2)*sizeof(*(h->xb)));
    if (h->xb == NULL) { printf("error: could not allocate memory for xb\n"); return -1; }
    h->JJ = (REAL *)malloc(3*params.nx*params.ny*params.nz*sizeof(*(h->JJ)));
    if (h->JJ == NULL) { printf("error: could not allocate memory for JJ\n"); return -1; }
    h->detJac = (REAL *)malloc((params.nx+2)*(params.ny+2)*(params.nz+2)*sizeof(*(h->detJac)));
    if (h->detJac == NULL) { printf("error: could not allocate memory for detJac\n"); return -1; }
    if (params.dumpCellVol == 1) {
        h->cellVol = (REAL *)malloc(params.nx*params.ny*params.nz*sizeof(*(h->cellVol)));
        if (h->cellVol == NULL) { printf("error: could not allocate memory for cellVol\n"); return -1; } }
    if ((params.dumpConvexity == 1) || (params.redConvex == 1)) {
        h->convexity = (REAL *)malloc(params.nx*params.ny*params.nz*sizeof(*(h->convexity)));
        if (h->cellVol == NULL) { printf("error: could not allocate memory for convexity\n"); return -1; } }
    if ((params.dumpWedgeMin == 1) || (params.redWedgeMin == 1)) {
        h->wedgeMin = (REAL *)malloc(params.nx*params.ny*params.nz*sizeof(*(h->wedgeMin)));
        if (h->wedgeMin == NULL) { printf("error: could not allocate memory for wedgeMin\n"); return -1; } }
    if (params.inertia == true) {
        h->uu = (REAL *)malloc(3*params.nx*params.ny*params.nz*sizeof(*(h->uu)));
        if (h->uu == NULL) { printf("error: could not allocate memory for uu\n"); return -1; } }

    // Allocate global memory on GPU device.
    errCuda = hipMalloc((void**)&(d->B0), 3*(params.nx+2)*(params.ny+2)*(params.nz+2)*sizeof(*(d->B0)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for B0\n"); exit(EXIT_FAILURE); }
    errCuda = hipMalloc((void**)&(d->B), 3*(params.nx+2)*(params.ny+2)*(params.nz+2)*sizeof(*(d->B)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for B\n"); exit(EXIT_FAILURE); }
    errCuda = hipMalloc((void**)&(d->xb), 3*(params.nx+2)*(params.ny+2)*(params.nz+2)*sizeof(*(d->xb)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for xb\n"); exit(EXIT_FAILURE); }
    errCuda = hipMalloc((void**)&(d->J), 3*params.nx*params.ny*params.nz*sizeof(*(d->J)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for J\n"); exit(EXIT_FAILURE); }
    errCuda = hipMalloc((void**)&(d->xb_new), 3*(params.nx+2)*(params.ny+2)*(params.nz+2)*sizeof(*(d->xb_new)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for xb_new\n"); exit(EXIT_FAILURE); }
    errCuda = hipMalloc((void**)&(d->xb_tmp), 3*(params.nx+2)*(params.ny+2)*(params.nz+2)*sizeof(*(d->xb_tmp)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for xb_tmp\n"); exit(EXIT_FAILURE); }
    if (params.inertia == false) {
        errCuda = hipMalloc((void**)&(d->kk), 3*params.nx*params.ny*params.nz*6*sizeof(*(d->kk)));
        if (hipSuccess != errCuda) { printf("error: could not allocate device memory for kk\n"); exit(EXIT_FAILURE); } }
    else {
        errCuda = hipMalloc((void**)&(d->kk), 6*params.nx*params.ny*params.nz*6*sizeof(*(d->kk)));
        if (hipSuccess != errCuda) { printf("error: could not allocate device memory for kk\n"); exit(EXIT_FAILURE); } }
    if (params.pressure == true) {
        errCuda = hipMalloc((void**)&(d->gradP), 3*params.nx*params.ny*params.nz*sizeof(*(d->gradP)));
        if (hipSuccess != errCuda) { printf("error: could not allocate device memory for gradP\n"); exit(EXIT_FAILURE); } }
    errCuda = hipMalloc((void**)&(d->maxDelta), gridSize[0]*gridSize[1]*gridSize[2]*sizeof(*(d->maxDelta)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for maxDelta\n"); exit(EXIT_FAILURE); }
    errCuda = hipMalloc((void**)&(d->detJac), (params.nx+2)*(params.ny+2)*(params.nz+2)*sizeof(*(d->detJac)));
    if (hipSuccess != errCuda) { printf("error: could not allocate device memory for detJac\n"); exit(EXIT_FAILURE); }
    if (params.inertia == true) {
        errCuda = hipMalloc((void**)&(d->uu), 3*params.nx*params.ny*params.nz*sizeof(*(d->uu)));
        if (hipSuccess != errCuda) { printf("error: could not allocate device memory for uu\n"); exit(EXIT_FAILURE); }
        errCuda = hipMalloc((void**)&(d->uu_new), 3*params.nx*params.ny*params.nz*sizeof(*(d->uu_new)));
        if (hipSuccess != errCuda) { printf("error: could not allocate device memory for uu_new\n"); exit(EXIT_FAILURE); }
        errCuda = hipMalloc((void**)&(d->uu_tmp), 3*params.nx*params.ny*params.nz*sizeof(*(d->uu_tmp)));
        if (hipSuccess != errCuda) { printf("error: could not allocate device memory for uu_tmp\n"); exit(EXIT_FAILURE); }
    }

    return 0;
}


// Free host and device memory.
int freeMemory(struct VarsHost *h, struct VarsDev *d, struct Parameters params) {
    free(h->B0); free(h->B); free(h->JJ); free(h->xb); free(h->detJac);
    hipFree(d->kk); hipFree(d->xb_new); hipFree(d->maxDelta); hipFree(d->xb_tmp);
    hipFree(d->detJac);
    hipFree(d->xb); hipFree(d->B0); hipFree(d->B); hipFree(d->J);
    if (params.dumpCellVol == 1)
        free(h->cellVol);
    if ((params.dumpConvexity == 1) || (params.redConvex == 1))
        free(h->convexity);
    if ((params.dumpWedgeMin == 1) || (params.redWedgeMin == 1))
        free(h->wedgeMin);
    if (params.inertia == true) {
        free(h->uu);
        hipFree(d->uu); hipFree(d->uu_new); hipFree(d->uu_tmp);
    }

    return 0;
}

