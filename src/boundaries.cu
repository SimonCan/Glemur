#include "hip/hip_runtime.h"
// boundaries.cu
//
// Boundary routines for host and device arrays.
//

#include "boundaries.h"
#include "maths.h"

// in case of a distorted initial grid we need to add the distortion to the "second" ghost cells
__device__ REAL addDist(int i, int j, REAL z, int l)
{
    REAL x, y;
    REAL distX, distY;

    x = dev_p.dx*i + dev_p.Ox;
    y = dev_p.dy*j + dev_p.Oy;

    distX = 0;
    distY = 0;

    if (dev_p.initDistCode == 0) {
        distY = -dev_p.initShearA * sin(dev_p.initShearK*2*PI*(x+dev_p.Ox-dev_p.dx/2)/(dev_p.Lx+dev_p.dx)) * z;
        distX = -dev_p.initShearB * sin(dev_p.initShearK*2*PI*(distY+y+dev_p.Oy-dev_p.dy/2)/(dev_p.Ly+dev_p.dy)) * z;
    }
    if (dev_p.initDistCode == 1) {
        distX = 0;
        distY = dev_p.initShearA * exp(-dev_p.initShearK*x*x)*(1-y*y)*(exp(-1*z+1) - 0.15366);
    }

    if (l == 0)
        return distX;
    else
        return distY;
}


// update B and detJac at the boundary (previously done within B_JacB0 (inefficient))
__global__ void updateBbound(struct varsDev_t d, int face)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int p = threadIdx.x;
    int q = threadIdx.y;
    int b, l, side;
    REAL Bx, By, Bz, detJac1;      // local variables (registers)

    __shared__ REAL xbs[3][3][18][18], B0s[3][16][16], jacs[3][3][16][16];

    // xy faces
    if (face == 2) {
        for (side = 0; side < 2; side++) {
            if (dev_p.zPeri == false) {
                if ((i < dev_p.nx) && (j < dev_p.ny)) {
                    // copy from global memory to shared memory for faster computation
                    for (l = 0; l < 3; l++) {
                        for (b = 0; b < 2; b++)
                            xbs[l][b+1-side][p+1][q+1] = d.xb[l + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + (b+side*dev_p.nz)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        B0s[l][p][q] = d.B0[l + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + side*(dev_p.nz+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                    }
                }
                __syncthreads();
                if ((i < dev_p.nx) && (j < dev_p.ny)) {
                    for (l = 0; l < 2; l++)
                        xbs[l][0+2*side][p+1][q+1] = xbs[l][1][p+1][q+1] - addDist(i, j, xbs[2][1][p+1][q+1], l)
                            + addDist(i, j, 2*dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, l);
                    xbs[2][0+2*side][p+1][q+1] = xbs[2][1][p+1][q+1] + (2*side-1)*dev_p.dz;
                }
                __syncthreads();
                if ((i < dev_p.nx) && (j < dev_p.ny)) {
                    // get xbs at the edges
                    if (p == 0) {
                        xbs[0][1][0][q+1]  = xbs[0][1][1][q+1] - dev_p.dx - addDist(i, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 0) +
                            addDist(i-1, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 0);
                        xbs[1][1][0][q+1]  = xbs[1][1][1][q+1] - addDist(i, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 1) +
                            addDist(i-1, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 1);
                        xbs[2][1][0][q+1]  = xbs[2][1][1][q+1];
                        xbs[0][1][17][q+1] = xbs[0][1][16][q+1] + dev_p.dx - addDist(i+15, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 0) +
                            addDist(i+16, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 0);
                        xbs[1][1][17][q+1] = xbs[1][1][16][q+1] - addDist(i+15, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 1) +
                            addDist(i+16, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 1);
                        xbs[2][1][17][q+1] = xbs[2][1][16][q+1];
                    }
                    if (i == dev_p.nx-1) {
                        xbs[0][1][p+2][q+1] = xbs[0][1][p+1][q+1] + dev_p.dx - addDist(i, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 0) +
                            addDist(i+1, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 0);
                        xbs[1][1][p+2][q+1] = xbs[1][1][p+1][q+1] - addDist(i, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 1) +
                            addDist(i+1, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 1);
                        xbs[2][1][p+2][q+1] = xbs[2][1][p+1][q+1];
                    }
                    if (q == 0) {
                        xbs[0][1][p+1][0]  = xbs[0][1][p+1][1] - addDist(i, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 0) +
                            addDist(i, j-1, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 0);
                        xbs[1][1][p+1][0]  = xbs[1][1][p+1][1] - dev_p.dy - addDist(i, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 1) +
                            addDist(i, j-1, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 1);
                        xbs[2][1][p+1][0]  = xbs[2][1][p+1][1];
                        xbs[0][1][p+1][17] = xbs[0][1][p+1][16] - addDist(i, j+15, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 0) +
                            addDist(i, j+16, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 0);
                        xbs[1][1][p+1][17] = xbs[1][1][p+1][16] + dev_p.dy - addDist(i, j+15, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 1) +
                            addDist(i, j+16, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 1);
                        xbs[2][1][p+1][17] = xbs[2][1][p+1][16];
                    }
                    if (j == dev_p.ny-1) {
                        xbs[0][1][p+1][q+2] = xbs[0][1][p+1][q+1] - addDist(i, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 0) +
                            addDist(i, j+1, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 0);
                        xbs[1][1][p+1][q+2] = xbs[1][1][p+1][q+1] + dev_p.dy - addDist(i, j, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 1) +
                            addDist(i, j+1, dev_p.dz*(2*side-1) + dev_p.Oz + dev_p.Lz*side, 1);
                        xbs[2][1][p+1][q+2] = xbs[2][1][p+1][q+1];
                    }
                }
            }
            else {
                if ((i < dev_p.nx) && (j < dev_p.ny)) {
                    // copy from global memory to shared memory for faster computation
                    if (side == 0) {
                        for (l = 0; l < 2; l++) {
                            xbs[l][0][p+1][q+1] = d.xb[l + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + (dev_p.nz-1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                            xbs[l][1][p+1][q+1] = d.xb[l + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + (dev_p.nz+0)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                            xbs[l][2][p+1][q+1] = d.xb[l + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + 1*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        }
                        xbs[2][0][p+1][q+1] = d.xb[2 + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + (dev_p.nz-1)*(dev_p.nx+2)*(dev_p.ny+2)*3] - dev_p.Lz - dev_p.dz;
                        xbs[2][1][p+1][q+1] = d.xb[2 + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + (dev_p.nz+0)*(dev_p.nx+2)*(dev_p.ny+2)*3] - dev_p.Lz - dev_p.dz;
                        xbs[2][2][p+1][q+1] = d.xb[2 + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + 1*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        for (l = 0; l < 3; l++)
                            B0s[l][p][q] = d.B0[l + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + (dev_p.nz+0)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                    }
                    else {
                        for (l = 0; l < 2; l++) {
                            xbs[l][0][p+1][q+1] = d.xb[l + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + (dev_p.nz+0)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                            xbs[l][1][p+1][q+1] = d.xb[l + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + 1*(dev_p.nx+2)*(dev_p.ny+2)*3];
                            xbs[l][2][p+1][q+1] = d.xb[l + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + 2*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        }
                        xbs[2][0][p+1][q+1] = d.xb[2 + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + (dev_p.nz+0)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        xbs[2][1][p+1][q+1] = d.xb[2 + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + 1*(dev_p.nx+2)*(dev_p.ny+2)*3] + dev_p.Lz + dev_p.dz;
                        xbs[2][2][p+1][q+1] = d.xb[2 + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + 2*(dev_p.nx+2)*(dev_p.ny+2)*3] + dev_p.Lz + dev_p.dz;
                        for (l = 0; l < 3; l++)
                            B0s[l][p][q] = d.B0[l + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + 1*(dev_p.nx+2)*(dev_p.ny+2)*3];
                    }
                }
                __syncthreads();
                if ((i < dev_p.nx) && (j < dev_p.ny)) {
                    // get xbs at the edges
                    for (l = 0; l < 2; l++) {
                        if (p == 0)
                            xbs[l][1][0][q+1]  = d.xb[l + (i+0)*3 + (j+1)*(dev_p.nx+2)*3 + (side+(1-side)*dev_p.nz)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        if ((p == blockDim.x-1) || (i == dev_p.nx-1))
                            xbs[l][1][p+2][q+1] = d.xb[l + (i+2)*3 + (j+1)*(dev_p.nx+2)*3 + (side+(1-side)*dev_p.nz)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        if (q == 0)
                            xbs[l][1][p+1][0]  = d.xb[l + (i+1)*3 + (j+0)*(dev_p.nx+2)*3 + (side+(1-side)*dev_p.nz)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        if ((q == blockDim.y-1) || (j == dev_p.ny-1))
                            xbs[l][1][p+1][q+2] = d.xb[l + (i+1)*3 + (j+2)*(dev_p.nx+2)*3 + (side+(1-side)*dev_p.nz)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                    }
                    if (p == 0)
                        xbs[2][1][0][q+1]  = d.xb[2 + (i+0)*3 + (j+1)*(dev_p.nx+2)*3 + (side+(1-side)*dev_p.nz)*(dev_p.nx+2)*(dev_p.ny+2)*3] + (2*side-1)*(dev_p.Lz + dev_p.dz);
                    if ((p == blockDim.x-1) || (i == dev_p.nx-1))
                        xbs[2][1][p+2][q+1] = d.xb[2 + (i+2)*3 + (j+1)*(dev_p.nx+2)*3 + (side+(1-side)*dev_p.nz)*(dev_p.nx+2)*(dev_p.ny+2)*3] + (2*side-1)*(dev_p.Lz + dev_p.dz);
                    if (q == 0)
                        xbs[2][1][p+1][0]  = d.xb[2 + (i+1)*3 + (j+0)*(dev_p.nx+2)*3 + (side+(1-side)*dev_p.nz)*(dev_p.nx+2)*(dev_p.ny+2)*3] + (2*side-1)*(dev_p.Lz + dev_p.dz);
                    if ((q == blockDim.y-1) || (j == dev_p.ny-1))
                        xbs[2][1][p+1][q+2] = d.xb[2 + (i+1)*3 + (j+2)*(dev_p.nx+2)*3 + (side+(1-side)*dev_p.nz)*(dev_p.nx+2)*(dev_p.ny+2)*3] + (2*side-1)*(dev_p.Lz + dev_p.dz);
                }
            }
            __syncthreads();
            if ((i < dev_p.nx) && (j < dev_p.ny)) {
                for (l = 0; l < 3; l++) {
                    jacs[l][0][p][q] = (xbs[l][1][p+2][q+1] - xbs[l][1][p+0][q+1]) * dev_p.dx1 / 2;
                    jacs[l][1][p][q] = (xbs[l][1][p+1][q+2] - xbs[l][1][p+1][q+0]) * dev_p.dy1 / 2;
                    jacs[l][2][p][q] = (xbs[l][2][p+1][q+1] - xbs[l][0][p+1][q+1]) * dev_p.dz1 / 2;
                }
                detJac1 = 1/(jacs[0][0][p][q]*jacs[1][1][p][q]*jacs[2][2][p][q] + jacs[0][1][p][q]*jacs[1][2][p][q]*jacs[2][0][p][q] + jacs[0][2][p][q]*jacs[1][0][p][q]*jacs[2][1][p][q] -
                             jacs[0][0][p][q]*jacs[1][2][p][q]*jacs[2][1][p][q] - jacs[0][1][p][q]*jacs[1][0][p][q]*jacs[2][2][p][q] - jacs[0][2][p][q]*jacs[1][1][p][q]*jacs[2][0][p][q]);

                // compute the magnetic field from the original one
                Bx = (jacs[0][0][p][q]*B0s[0][p][q] + jacs[0][1][p][q]*B0s[1][p][q] + jacs[0][2][p][q]*B0s[2][p][q])*detJac1;
                By = (jacs[1][0][p][q]*B0s[0][p][q] + jacs[1][1][p][q]*B0s[1][p][q] + jacs[1][2][p][q]*B0s[2][p][q])*detJac1;
                if (dev_p.zPeri == true)
                    Bz = (jacs[2][0][p][q]*B0s[0][p][q] + jacs[2][1][p][q]*B0s[1][p][q] + jacs[2][2][p][q]*B0s[2][p][q])*detJac1;
                else
                    Bz = B0s[2][p][q];  // set B.n to B0.n on the boundary

                // copy back to global memory
                d.B[0 + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + side*(dev_p.nz+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] = Bx;
                d.B[1 + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + side*(dev_p.nz+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] = By;
                d.B[2 + (i+1)*3 + (j+1)*(dev_p.nx+2)*3 + side*(dev_p.nz+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] = Bz;
                d.detJac[(i+1) + (j+1)*(dev_p.nx+2) + side*(dev_p.nz+1)*(dev_p.nx+2)*(dev_p.ny+2)] = 1/detJac1;
            }
            __syncthreads();
        }
    }

    // xz faces
    if (face == 1) {
        for (side = 0; side < 2; side++) {
            if (dev_p.yPeri == false) {
                if ((i < dev_p.nx) && (j < dev_p.nz)) {
                    // copy from global memory to shared memory for faster computation
                    for (l = 0; l < 3; l++) {
                        for (b = 0; b < 2; b++)
                            xbs[l][b+1-side][p+1][q+1] = d.xb[l + (i+1)*3 + (b+side*dev_p.ny)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        B0s[l][p][q] = d.B0[l + (i+1)*3 + side*(dev_p.ny+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                    }
                }
                __syncthreads();
                if ((i < dev_p.nx) && (j < dev_p.nz)) {
                    for (l = 0; l < 3; l += 2)
                        xbs[l][0+2*side][p+1][q+1] = xbs[l][1][p+1][q+1];
                    xbs[1][0+2*side][p+1][q+1] = xbs[1][1][p+1][q+1] + (2*side-1)*dev_p.dy;
                }
                __syncthreads();
                if ((i < dev_p.nx) && (j < dev_p.nz)) {
                    // get xbs at the edges
                    if (p == 0) {
                        xbs[0][1][0][q+1]  = xbs[0][1][1][q+1] - dev_p.dx;
                        xbs[1][1][0][q+1]  = xbs[1][1][1][q+1];
                        xbs[2][1][0][q+1]  = xbs[2][1][1][q+1];
                        xbs[0][1][17][q+1] = xbs[0][1][16][q+1] + dev_p.dx;
                        xbs[1][1][17][q+1] = xbs[1][1][16][q+1];
                        xbs[2][1][17][q+1] = xbs[2][1][16][q+1];
                    }
                    if (i == dev_p.nx-1) {
                        xbs[0][1][p+2][q+1] = xbs[0][1][p+1][q+1] + dev_p.dx;
                        xbs[1][1][p+2][q+1] = xbs[1][1][p+1][q+1];
                        xbs[2][1][p+2][q+1] = xbs[2][1][p+1][q+1];
                    }
                    if (q == 0) {
                        xbs[0][1][p+1][0]  = xbs[0][1][p+1][1];
                        xbs[1][1][p+1][0]  = xbs[1][1][p+1][1];
                        xbs[2][1][p+1][0]  = xbs[2][1][p+1][1] - dev_p.dz;
                        xbs[0][1][p+1][17] = xbs[0][1][p+1][16];
                        xbs[1][1][p+1][17] = xbs[1][1][p+1][16];
                        xbs[2][1][p+1][17] = xbs[2][1][p+1][16] + dev_p.dz;
                    }
                    if (j == dev_p.nz-1) {
                        xbs[0][1][p+1][q+2] = xbs[0][1][p+1][q+1];
                        xbs[1][1][p+1][q+2] = xbs[1][1][p+1][q+1];
                        xbs[2][1][p+1][q+2] = xbs[2][1][p+1][q+1] + dev_p.dz;
                    }
                }
            }
            else {
                if ((i < dev_p.nx) && (j < dev_p.nz)) {
                    // copy from global memory to shared memory for faster computation
                    if (side == 0) {
                        for (l = 0; l < 3; l += 2) {
                            xbs[l][0][p+1][q+1] = d.xb[l + (i+1)*3 + (dev_p.ny-1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                            xbs[l][1][p+1][q+1] = d.xb[l + (i+1)*3 + (dev_p.ny+0)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                            xbs[l][2][p+1][q+1] = d.xb[l + (i+1)*3 + 1*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        }
                        xbs[1][0][p+1][q+1] = d.xb[1 + (i+1)*3 + (dev_p.ny-1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] - dev_p.Ly - dev_p.dy;
                        xbs[1][1][p+1][q+1] = d.xb[1 + (i+1)*3 + (dev_p.ny+0)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] - dev_p.Ly - dev_p.dy;
                        xbs[1][2][p+1][q+1] = d.xb[1 + (i+1)*3 + 1*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        for (l = 0; l < 3; l++)
                            B0s[l][p][q] = d.B0[l + (i+1)*3 + (dev_p.ny+0)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                    }
                    else {
                        for (l = 0; l < 3; l += 2) {
                            xbs[l][0][p+1][q+1] = d.xb[l + (i+1)*3 + (dev_p.ny+0)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                            xbs[l][1][p+1][q+1] = d.xb[l + (i+1)*3 + 1*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                            xbs[l][2][p+1][q+1] = d.xb[l + (i+1)*3 + 2*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        }
                        xbs[1][0][p+1][q+1] = d.xb[1 + (i+1)*3 + (dev_p.ny+0)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        xbs[1][1][p+1][q+1] = d.xb[1 + (i+1)*3 + 1*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] + dev_p.Ly + dev_p.dy;
                        xbs[1][2][p+1][q+1] = d.xb[1 + (i+1)*3 + 2*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] + dev_p.Ly + dev_p.dy;
                        for (l = 0; l < 3; l++)
                            B0s[l][p][q] = d.B0[l + (i+1)*3 + 1*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                    }
                }
                __syncthreads();
                if ((i < dev_p.nx) && (j < dev_p.nz)) {
                    // get xbs at the edges
                    for (l = 0; l < 3; l += 2) {
                        if (p == 0)
                            xbs[l][1][0][q+1]  = d.xb[l + (i+0)*3 + (side+(1-side)*dev_p.ny)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        if ((p == blockDim.x-1) || (i == dev_p.nx-1))
                            xbs[l][1][p+2][q+1] = d.xb[l + (i+2)*3 + (side+(1-side)*dev_p.ny)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        if (q == 0)
                            xbs[l][1][p+1][0]  = d.xb[l + (i+1)*3 + (side+(1-side)*dev_p.ny)*(dev_p.nx+2)*3 + (j+0)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        if ((q == blockDim.z-1) || (j == dev_p.nz-1))
                            xbs[l][1][p+1][q+2] = d.xb[l + (i+1)*3 + (side+(1-side)*dev_p.ny)*(dev_p.nx+2)*3 + (j+2)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                    }
                    if (p == 0)
                        xbs[1][1][0][q+1]  = d.xb[1 + (i+0)*3 + (side+(1-side)*dev_p.ny)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] + (2*side-1)*(dev_p.Ly + dev_p.dy);
                    if ((p == blockDim.x-1) || (i == dev_p.nx-1))
                        xbs[1][1][p+2][q+1] = d.xb[1 + (i+2)*3 + (side+(1-side)*dev_p.ny)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] + (2*side-1)*(dev_p.Ly + dev_p.dy);
                    if (q == 0)
                        xbs[1][1][p+1][0]  = d.xb[1 + (i+1)*3 + (side+(1-side)*dev_p.ny)*(dev_p.nx+2)*3 + (j+0)*(dev_p.nx+2)*(dev_p.ny+2)*3] + (2*side-1)*(dev_p.Ly + dev_p.dy);
                    if ((q == blockDim.z-1) || (j == dev_p.nz-1))
                        xbs[1][1][p+1][q+2] = d.xb[1 + (i+1)*3 + (side+(1-side)*dev_p.ny)*(dev_p.nx+2)*3 + (j+2)*(dev_p.nx+2)*(dev_p.ny+2)*3] + (2*side-1)*(dev_p.Ly + dev_p.dy);
                }
            }
            __syncthreads();
            if ((i < dev_p.nx) && (j < dev_p.nz)) {
                for (l = 0; l < 3; l++) {
                    jacs[l][0][p][q] = (xbs[l][1][p+2][q+1] - xbs[l][1][p+0][q+1]) * dev_p.dx1 / 2;
                    jacs[l][1][p][q] = (xbs[l][2][p+1][q+1] - xbs[l][0][p+1][q+1]) * dev_p.dy1 / 2;
                    jacs[l][2][p][q] = (xbs[l][1][p+1][q+2] - xbs[l][1][p+1][q+0]) * dev_p.dz1 / 2;
                }
                detJac1 = 1/(jacs[0][0][p][q]*jacs[1][1][p][q]*jacs[2][2][p][q] + jacs[0][1][p][q]*jacs[1][2][p][q]*jacs[2][0][p][q] + jacs[0][2][p][q]*jacs[1][0][p][q]*jacs[2][1][p][q] -
                             jacs[0][0][p][q]*jacs[1][2][p][q]*jacs[2][1][p][q] - jacs[0][1][p][q]*jacs[1][0][p][q]*jacs[2][2][p][q] - jacs[0][2][p][q]*jacs[1][1][p][q]*jacs[2][0][p][q]);

                // compute the magnetic field from the original one
                Bx = (jacs[0][0][p][q]*B0s[0][p][q] + jacs[0][1][p][q]*B0s[1][p][q] + jacs[0][2][p][q]*B0s[2][p][q])*detJac1;
                if (dev_p.yPeri == true)
                    By = (jacs[1][0][p][q]*B0s[0][p][q] + jacs[1][1][p][q]*B0s[1][p][q] + jacs[1][2][p][q]*B0s[2][p][q])*detJac1;
                else
                    By = B0s[1][p][q];    // set B.n to B0.n on the boundary
                Bz = (jacs[2][0][p][q]*B0s[0][p][q] + jacs[2][1][p][q]*B0s[1][p][q] + jacs[2][2][p][q]*B0s[2][p][q])*detJac1;

                // copy back to global memory
                d.B[0 + (i+1)*3 + side*(dev_p.ny+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] = Bx;
                d.B[1 + (i+1)*3 + side*(dev_p.ny+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] = By;
                d.B[2 + (i+1)*3 + side*(dev_p.ny+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] = Bz;
                d.detJac[(i+1) + side*(dev_p.ny+1)*(dev_p.nx+2) + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)] = 1/detJac1;
            }
            __syncthreads();
        }
    }

    // yz faces
    if (face == 0) {
        for (side = 0; side < 2; side++) {
            if (dev_p.xPeri == false) {
                if ((i < dev_p.ny) && (j < dev_p.nz)) {
                    // copy from global memory to shared memory for faster computation
                    for (l = 0; l < 3; l++) {
                        for (b = 0; b < 2; b++)
                            xbs[l][b+1-side][p+1][q+1] = d.xb[l + (b+side*dev_p.nx)*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        B0s[l][p][q] = d.B0[l + side*(dev_p.nx+1)*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                    }
                }
                __syncthreads();
                if ((i < dev_p.ny) && (j < dev_p.nz)) {
                    for (l = 1; l < 3; l++)
                        xbs[l][0+2*side][p+1][q+1] = xbs[l][1][p+1][q+1];
                    xbs[0][0+2*side][p+1][q+1] = xbs[0][1][p+1][q+1] + (2*side-1)*dev_p.dx;
                }
                __syncthreads();
                if ((i < dev_p.ny) && (j < dev_p.nz)) {
                    // get xbs at the edges
                    if (p == 0) {
                        xbs[0][1][0][q+1]  = xbs[0][1][1][q+1];
                        xbs[1][1][0][q+1]  = xbs[1][1][1][q+1] - dev_p.dy;
                        xbs[2][1][0][q+1]  = xbs[2][1][1][q+1];
                        xbs[0][1][17][q+1] = xbs[0][1][16][q+1];
                        xbs[1][1][17][q+1] = xbs[1][1][16][q+1] + dev_p.dy;
                        xbs[2][1][17][q+1] = xbs[2][1][16][q+1];
                    }
                    if (i == dev_p.ny-1) {
                        xbs[0][1][p+2][q+1] = xbs[0][1][p+1][q+1];
                        xbs[1][1][p+2][q+1] = xbs[1][1][p+1][q+1] + dev_p.dy;
                        xbs[2][1][p+2][q+1] = xbs[2][1][p+1][q+1];
                    }
                    if (q == 0) {
                        xbs[0][1][p+1][0]  = xbs[0][1][p+1][1];
                        xbs[1][1][p+1][0]  = xbs[1][1][p+1][1];
                        xbs[2][1][p+1][0]  = xbs[2][1][p+1][1] - dev_p.dz;
                        xbs[0][1][p+1][17] = xbs[0][1][p+1][16];
                        xbs[1][1][p+1][17] = xbs[1][1][p+1][16];
                        xbs[2][1][p+1][17] = xbs[2][1][p+1][16] + dev_p.dz;
                    }
                    if (j == dev_p.nz-1) {
                        xbs[0][1][p+1][q+2] = xbs[0][1][p+1][q+1];
                        xbs[1][1][p+1][q+2] = xbs[1][1][p+1][q+1];
                        xbs[2][1][p+1][q+2] = xbs[2][1][p+1][q+1] + dev_p.dz;
                    }
                }
            }
            else {
                if ((i < dev_p.ny) && (j < dev_p.nz)) {
                    // copy from global memory to shared memory for faster computation
                    if (side == 0) {
                        for (l = 1; l < 3; l++) {
                            xbs[l][0][p+1][q+1] = d.xb[l + (dev_p.nx-1)*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                            xbs[l][1][p+1][q+1] = d.xb[l + (dev_p.nx+0)*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                            xbs[l][2][p+1][q+1] = d.xb[l + 1*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        }
                        xbs[0][0][p+1][q+1] = d.xb[0 + (dev_p.nx-1)*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] - dev_p.Lx - dev_p.dx;
                        xbs[0][1][p+1][q+1] = d.xb[0 + (dev_p.nx+0)*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] - dev_p.Lx - dev_p.dx;
                        xbs[0][2][p+1][q+1] = d.xb[0 + 1*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        for (l = 0; l < 3; l++)
                            B0s[l][p][q] = d.B0[l + (dev_p.nx+0)*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                    }
                    else {
                        for (l = 1; l < 3; l++) {
                            xbs[l][0][p+1][q+1] = d.xb[l + (dev_p.nx+0)*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                            xbs[l][1][p+1][q+1] = d.xb[l + 1*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                            xbs[l][2][p+1][q+1] = d.xb[l + 2*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        }
                        xbs[0][0][p+1][q+1] = d.xb[0 + (dev_p.nx+0)*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        xbs[0][1][p+1][q+1] = d.xb[0 + 1*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] + dev_p.Lx + dev_p.dx;
                        xbs[0][2][p+1][q+1] = d.xb[0 + 2*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] + dev_p.Lx + dev_p.dx;
                        for (l = 0; l < 3; l++)
                            B0s[l][p][q] = d.B0[l + 1*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                    }
                }
                __syncthreads();
                if ((i < dev_p.ny) && (j < dev_p.nz)) {
                    // get xbs at the edges
                    for (l = 1; l < 3; l++) {
                        if (p == 0)
                            xbs[l][1][0][q+1]  = d.xb[l + (side+(1-side)*dev_p.nx)*3 + (i+0)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        if ((p == blockDim.y-1) || (i == dev_p.ny-1))
                            xbs[l][1][p+2][q+1] = d.xb[l + (side+(1-side)*dev_p.nx)*3 + (i+2)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        if (q == 0)
                            xbs[l][1][p+1][0]  = d.xb[l + (side+(1-side)*dev_p.nx)*3 + (i+1)*(dev_p.nx+2)*3 + (j+0)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                        if ((q == blockDim.z-1) || (j == dev_p.nz-1))
                            xbs[l][1][p+1][q+2] = d.xb[l + (side+(1-side)*dev_p.nx)*3 + (i+1)*(dev_p.nx+2)*3 + (j+2)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                    }
                    if (p == 0)
                        xbs[0][1][0][q+1]  = d.xb[0 + (side+(1-side)*dev_p.nx)*3 + (i+0)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] + (2*side-1)*(dev_p.Lx + dev_p.dx);
                    if ((p == blockDim.y-1) || (i == dev_p.ny-1))
                        xbs[0][1][p+2][q+1] = d.xb[0 + (side+(1-side)*dev_p.nx)*3 + (i+2)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] + (2*side-1)*(dev_p.Lx + dev_p.dx);
                    if (q == 0)
                        xbs[0][1][p+1][0]  = d.xb[0 + (side+(1-side)*dev_p.nx)*3 + (i+1)*(dev_p.nx+2)*3 + (j+0)*(dev_p.nx+2)*(dev_p.ny+2)*3] + (2*side-1)*(dev_p.Lx + dev_p.dx);
                    if ((q == blockDim.z-1) || (j == dev_p.nz-1))
                        xbs[0][1][p+1][q+2] = d.xb[0 + (side+(1-side)*dev_p.nx)*3 + (i+1)*(dev_p.nx+2)*3 + (j+2)*(dev_p.nx+2)*(dev_p.ny+2)*3] + (2*side-1)*(dev_p.Lx + dev_p.dx);
                }
            }
            __syncthreads();
            if ((i < dev_p.ny) && (j < dev_p.nz)) {
                for (l = 0; l < 3; l++) {
                    jacs[l][0][p][q] = (xbs[l][2][p+1][q+1] - xbs[l][0][p+1][q+1]) * dev_p.dx1 / 2;
                    jacs[l][1][p][q] = (xbs[l][1][p+2][q+1] - xbs[l][1][p+0][q+1]) * dev_p.dy1 / 2;
                    jacs[l][2][p][q] = (xbs[l][1][p+1][q+2] - xbs[l][1][p+1][q+0]) * dev_p.dz1 / 2;
                }
                detJac1 = 1/(jacs[0][0][p][q]*jacs[1][1][p][q]*jacs[2][2][p][q] + jacs[0][1][p][q]*jacs[1][2][p][q]*jacs[2][0][p][q] + jacs[0][2][p][q]*jacs[1][0][p][q]*jacs[2][1][p][q] -
                             jacs[0][0][p][q]*jacs[1][2][p][q]*jacs[2][1][p][q] - jacs[0][1][p][q]*jacs[1][0][p][q]*jacs[2][2][p][q] - jacs[0][2][p][q]*jacs[1][1][p][q]*jacs[2][0][p][q]);

                // compute the magnetic field from the original one
                if (dev_p.xPeri == true)
                    Bx = (jacs[0][0][p][q]*B0s[0][p][q] + jacs[0][1][p][q]*B0s[1][p][q] + jacs[0][2][p][q]*B0s[2][p][q])*detJac1;
                else
                    Bx = B0s[0][p][q];    // set B.n to B0.n on the boundary
                By = (jacs[1][0][p][q]*B0s[0][p][q] + jacs[1][1][p][q]*B0s[1][p][q] + jacs[1][2][p][q]*B0s[2][p][q])*detJac1;
                Bz = (jacs[2][0][p][q]*B0s[0][p][q] + jacs[2][1][p][q]*B0s[1][p][q] + jacs[2][2][p][q]*B0s[2][p][q])*detJac1;

                // copy back to global memory
                d.B[0 + side*(dev_p.nx+1)*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] = Bx;
                d.B[1 + side*(dev_p.nx+1)*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] = By;
                d.B[2 + side*(dev_p.nx+1)*3 + (i+1)*(dev_p.nx+2)*3 + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] = Bz;
                d.detJac[side*(dev_p.nx+1) + (i+1)*(dev_p.nx+2) + (j+1)*(dev_p.nx+2)*(dev_p.ny+2)] = 1/detJac1;
            }
            __syncthreads();
        }
    }
}


// set vector field 'field' to be periodic
__global__ void setPeriFace(REAL *field, int face)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int l;

    if (face == 2) {
        if ((i < dev_p.nx+2) && (j < dev_p.ny+2)) {
            for (l = 0; l < 3; l++) {
                field[l + i*3 + j*(dev_p.nx+2)*3 + 0*(dev_p.nx+2)*(dev_p.ny+2)*3] =
                field[l + i*3 + j*(dev_p.nx+2)*3 + (dev_p.nz+0)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                field[l + i*3 + j*(dev_p.nx+2)*3 + (dev_p.nz+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] =
                field[l + i*3 + j*(dev_p.nx+2)*3 + 1*(dev_p.nx+2)*(dev_p.ny+2)*3];
            }
        }
    }

    if (face == 1) {
        if ((i < dev_p.nx+2) && (j < dev_p.nz+2)) {
            for (l = 0; l < 3; l++) {
                field[l + i*3 + 0*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] =
                field[l + i*3 + (dev_p.ny+0)*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3];
                field[l + i*3 + (dev_p.ny+1)*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] =
                field[l + i*3 + 1*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3];
            }
        }
    }

    if (face == 0) {
        if ((i < dev_p.ny+2) && (j < dev_p.nz+2)) {
            for (l = 0; l < 3; l++) {
                field[l + 0*3 + i*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] =
                field[l + (dev_p.nx+0)*3 + i*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3];
                field[l + (dev_p.nx+1)*3 + i*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] =
                field[l + 1*3 + i*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3];
            }
        }
    }
}


// set 'field' to be periodic (host code)
void setPeriHost(REAL *field, struct parameters_t p)
{
    int i, j, l;

    if (p.zPeri) {
        for (i = 0; i < p.nx+2; i++)
            for (j = 0; j < p.ny+2; j++)
                for (l = 0; l < 3; l++) {
                    field[l + i*3 + j*(p.nx+2)*3 + 0*(p.nx+2)*(p.ny+2)*3] =
                    field[l + i*3 + j*(p.nx+2)*3 + (p.nz+0)*(p.nx+2)*(p.ny+2)*3];
                    field[l + i*3 + j*(p.nx+2)*3 + (p.nz+1)*(p.nx+2)*(p.ny+2)*3] =
                    field[l + i*3 + j*(p.nx+2)*3 + 1*(p.nx+2)*(p.ny+2)*3];
                }
    }

    if (p.yPeri) {
        for (i = 0; i < p.nx+2; i++)
            for (j = 0; j < p.nz+2; j++)
                for (l = 0; l < 3; l++) {
                    field[l + i*3 + 0*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] =
                    field[l + i*3 + (p.ny+0)*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3];
                    field[l + i*3 + (p.ny+1)*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] =
                    field[l + i*3 + 1*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3];
                }
    }

    if (p.xPeri) {
        for (i = 0; i < p.ny+2; i++)
            for (j = 0; j < p.nz+2; j++)
                for (l = 0; l < 3; l++) {
                    field[l + 0*3 + i*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] =
                    field[l + (p.nx+0)*3 + i*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3];
                    field[l + (p.nx+1)*3 + i*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] =
                    field[l + 1*3 + i*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3];
                }
    }
}


// set the grid to be periodic
__global__ void setGridPeriFace(REAL *xb, int face)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int l;

    if (face == 2) {
        if ((i < dev_p.nx+2) && (j < dev_p.ny+2)) {
            for (l = 0; l < 2; l++) {
                xb[l + i*3 + j*(dev_p.nx+2)*3 + 0*(dev_p.nx+2)*(dev_p.ny+2)*3] =
                xb[l + i*3 + j*(dev_p.nx+2)*3 + (dev_p.nz+0)*(dev_p.nx+2)*(dev_p.ny+2)*3];
                xb[l + i*3 + j*(dev_p.nx+2)*3 + (dev_p.nz+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] =
                xb[l + i*3 + j*(dev_p.nx+2)*3 + 1*(dev_p.nx+2)*(dev_p.ny+2)*3];
            }
            xb[2 + i*3 + j*(dev_p.nx+2)*3 + 0*(dev_p.nx+2)*(dev_p.ny+2)*3] =
            xb[2 + i*3 + j*(dev_p.nx+2)*3 + (dev_p.nz+0)*(dev_p.nx+2)*(dev_p.ny+2)*3] - dev_p.Lz - dev_p.dz;
            xb[2 + i*3 + j*(dev_p.nx+2)*3 + (dev_p.nz+1)*(dev_p.nx+2)*(dev_p.ny+2)*3] =
            xb[2 + i*3 + j*(dev_p.nx+2)*3 + 1*(dev_p.nx+2)*(dev_p.ny+2)*3] + dev_p.Lz + dev_p.dz;
        }
    }

    if (face == 1) {
        if ((i < dev_p.nx+2) && (j < dev_p.nz+2)) {
            for (l = 0; l < 3; l += 2) {
                xb[l + i*3 + 0*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] =
                xb[l + i*3 + (dev_p.ny+0)*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3];
                xb[l + i*3 + (dev_p.ny+1)*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] =
                xb[l + i*3 + 1*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3];
            }
            xb[1 + i*3 + 0*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] =
            xb[1 + i*3 + (dev_p.ny+0)*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] - dev_p.Ly - dev_p.dy;
            xb[1 + i*3 + (dev_p.ny+1)*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] =
            xb[1 + i*3 + 1*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] + dev_p.Ly + dev_p.dy;
        }
    }

    if (face == 0) {
        if ((i < dev_p.ny+2) && (j < dev_p.nz+2)) {
            for (l = 1; l < 3; l++) {
                xb[l + 0*3 + i*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] =
                xb[l + (dev_p.nx+0)*3 + i*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3];
                xb[l + (dev_p.nx+1)*3 + i*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] =
                xb[l + 1*3 + i*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3];
            }
            xb[0 + 0*3 + i*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] =
            xb[0 + (dev_p.nx+0)*3 + i*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] - dev_p.Lx - dev_p.dx;
            xb[0 + (dev_p.nx+1)*3 + i*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] =
            xb[0 + 1*3 + i*(dev_p.nx+2)*3 + j*(dev_p.nx+2)*(dev_p.ny+2)*3] + dev_p.Lx + dev_p.dx;
        }
    }
}


// set the grid to be periodic (host code)
void setGridPeriHost(REAL *xb, struct parameters_t p)
{
    int i, j, l;

    if (p.zPeri) {
        for (i = 0; i < p.nx+2; i++)
            for (j = 0; j < p.ny+2; j++)
                for (l = 0; l < 2; l++) {
                    xb[l + i*3 + j*(p.nx+2)*3 + 0*(p.nx+2)*(p.ny+2)*3] =
                    xb[l + i*3 + j*(p.nx+2)*3 + (p.nz+0)*(p.nx+2)*(p.ny+2)*3];
                    xb[l + i*3 + j*(p.nx+2)*3 + (p.nz+1)*(p.nx+2)*(p.ny+2)*3] =
                    xb[l + i*3 + j*(p.nx+2)*3 + 1*(p.nx+2)*(p.ny+2)*3];
                }
                xb[2 + i*3 + j*(p.nx+2)*3 + 0*(p.nx+2)*(p.ny+2)*3] =
                xb[2 + i*3 + j*(p.nx+2)*3 + (p.nz+0)*(p.nx+2)*(p.ny+2)*3] - p.Lz - p.dz;
                xb[2 + i*3 + j*(p.nx+2)*3 + (p.nz+1)*(p.nx+2)*(p.ny+2)*3] =
                xb[2 + i*3 + j*(p.nx+2)*3 + 1*(p.nx+2)*(p.ny+2)*3] + p.Lz + p.dz;
    }

    if (p.yPeri) {
        for (i = 0; i < p.nx+2; i++)
            for (j = 0; j < p.nz+2; j++)
                for (l = 0; l < 3; l += 2) {
                    xb[l + i*3 + 0*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] =
                    xb[l + i*3 + (p.ny+0)*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3];
                    xb[l + i*3 + (p.ny+1)*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] =
                    xb[l + i*3 + 1*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3];
                }
                xb[1 + i*3 + 0*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] =
                xb[1 + i*3 + (p.ny+0)*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] - p.Ly - p.dy;
                xb[1 + i*3 + (p.ny+1)*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] =
                xb[1 + i*3 + 1*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] + p.Ly + p.dy;
    }

    if (p.xPeri) {
        for (i = 0; i < p.ny+2; i++)
            for (j = 0; j < p.nz+2; j++)
                for (l = 1; l < 3; l++) {
                    xb[l + 0*3 + i*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] =
                    xb[l + (p.nx+0)*3 + i*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3];
                    xb[l + (p.nx+1)*3 + i*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] =
                    xb[l + 1*3 + i*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3];
                }
                xb[0 + 0*3 + i*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] =
                xb[0 + (p.nx+0)*3 + i*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] - p.Lx - p.dx;
                xb[0 + (p.nx+1)*3 + i*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] =
                xb[0 + 1*3 + i*(p.nx+2)*3 + j*(p.nx+2)*(p.ny+2)*3] + p.Lx + p.dx;
    }
}


void setBbound(dim3 dimGrid2dXY, dim3 dimGrid2dXZ, dim3 dimGrid2dYZ, dim3 dimBlock2d, struct varsDev_t d)
{
    updateBbound<<<dimGrid2dXY, dimBlock2d>>>(d, 2); hipDeviceSynchronize();
    updateBbound<<<dimGrid2dXZ, dimBlock2d>>>(d, 1); hipDeviceSynchronize();
    updateBbound<<<dimGrid2dYZ, dimBlock2d>>>(d, 0); hipDeviceSynchronize();
}


void setPeri(dim3 dimGrid2dPlusXY, dim3 dimGrid2dPlusXZ, dim3 dimGrid2dPlusYZ, dim3 dimBlock2d, REAL *dev_field, struct parameters_t p)
{
    if (p.zPeri) setPeriFace<<<dimGrid2dPlusXY, dimBlock2d>>>(dev_field, 2);
    if (p.yPeri) setPeriFace<<<dimGrid2dPlusXZ, dimBlock2d>>>(dev_field, 1);
    if (p.xPeri) setPeriFace<<<dimGrid2dPlusYZ, dimBlock2d>>>(dev_field, 0);
}


void setGridPeri(dim3 dimGrid2dPlusXY, dim3 dimGrid2dPlusXZ, dim3 dimGrid2dPlusYZ, dim3 dimBlock2d, REAL *dev_xb, struct parameters_t p)
{
    if (p.zPeri) setGridPeriFace<<<dimGrid2dPlusXY, dimBlock2d>>>(dev_xb, 2);
    if (p.yPeri) setGridPeriFace<<<dimGrid2dPlusXZ, dimBlock2d>>>(dev_xb, 1);
    if (p.xPeri) setGridPeriFace<<<dimGrid2dPlusYZ, dimBlock2d>>>(dev_xb, 0);
}



