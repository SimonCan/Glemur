#include "hip/hip_runtime.h"
// init.cu
//
// Initial condition routines.
//

#include "global.h"
#include "init.h"
#include "blobsDomes.h"


// Set the residual magnetic energy for the corresponding configurations.
void initResiduals(struct Parameters p, struct Reduction *red)
{
    // Residual magnetic energy.
    if (strncmp(p.bInit, "Pontin09 ", 9) == 0)
        red->B2res = 1*p.Lx*p.Ly*p.Lz*p.ampl;
    if (strncmp(p.bInit, "analytic ", 9) == 0)
        red->B2res = 1*p.Lx*p.Ly*p.Lz*p.ampl;
    if (strncmp(p.bInit, "homZ ", 5) == 0)
        red->B2res = 1*p.Lx*p.Ly*p.Lz*p.ampl;
    if (strncmp(p.bInit, "blobs ", 6) == 0)
        red->B2res = 1*p.Lx*p.Ly*p.Lz*p.ampl;
    if (strncmp(p.bInit, "Borromean ", 10) == 0)
        red->B2res = 1*p.Lx*p.Ly*p.Lz*p.bGround;
}


// Create the initial magnetic field B0, the initial grid xb and initial velocity (if needed).
int initState(struct VarsHost h, struct Parameters p, struct Reduction *red)
{
    int  i, j, k, l, b;
    REAL x[p.nx+2], y[p.ny+2], z[p.nz+2];
    REAL tmp, r;

    //
    // magnetic field B0
    //

    if (strncmp(p.bInit, "blobsDome ", 10) == 0)
        initBlobsDome(h, p);
    if (strncmp(p.bInit, "blobsDomeShort ", 15) == 0)
        initBlobsDomeShort(h, p);
    if (strncmp(p.bInit, "blobsDomes2 ", 12) == 0)
        initBlobsDomes2(h, p);

    // Include the boundaries.
    for (k = 0; k < p.nz+2; k++) {
        z[k] = p.dz*(k-1) + p.Oz;
        for (j = 0; j < p.ny+2; j++) {
            y[j] = p.dy*(j-1) + p.Oy;
            for (i = 0; i < p.nx+2; i++) {
                x[i] = p.dx*(i-1) + p.Ox;
                // from [1]
                if (strncmp(p.bInit, "Pontin09 ", 9) == 0) {
                    // Field used by Pontin (2009).
                    h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] =
                        -2*p.ampl/(PI*p.ar) * y[j] *
                        (p.phi1*exp(-(x[i]*x[i]+y[j]*y[j])/(p.ar*p.ar)-(z[k]-p.L1)*(z[k]-p.L1)/(p.az*p.az)) +
                         p.phi2*exp(-(x[i]*x[i]+y[j]*y[j])/(p.ar*p.ar)-(z[k]-p.L2)*(z[k]-p.L2)/(p.az*p.az)));
                    h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] =
                        2*p.ampl/(PI*p.ar) * x[i] *
                        (p.phi1*exp(-(x[i]*x[i]+y[j]*y[j])/(p.ar*p.ar)-(z[k]-p.L1)*(z[k]-p.L1)/(p.az*p.az)) +
                         p.phi2*exp(-(x[i]*x[i]+y[j]*y[j])/(p.ar*p.ar)-(z[k]-p.L2)*(z[k]-p.L2)/(p.az*p.az)));
                    h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl;
                }

                // Field for which we know the relaxed state analytically.
                if (strncmp(p.bInit, "analytic ", 9) == 0) {
                    // Field used for testing.
                    h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] =
                        2*p.ampl*z[k]*exp(-(x[i]*x[i]+y[j]*y[j])/(p.ar*p.ar)-z[k]*z[k]/(p.az*p.az))*p.phi1/(p.az*p.az)*y[j];
                    h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] =
                        -2*p.ampl*z[k]*exp(-(x[i]*x[i]+y[j]*y[j])/(p.ar*p.ar)-z[k]*z[k]/(p.az*p.az))*p.phi1/(p.az*p.az)*x[i];
                    h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl;
                }

                // Field for which we know the relaxed state analytically extending in X.
                if (strncmp(p.bInit, "analyticX ", 10) == 0) {
                    // Field used for testing.
                    h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl;
                    h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] =
                        -2*p.ampl*x[i]*exp(-(z[k]*z[k]+y[j]*y[j])/(p.ar*p.ar)-x[i]*x[i]/(p.az*p.az))*p.phi1/(p.az*p.az)*z[k];
                    h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] =
                        2*p.ampl*x[i]*exp(-(z[k]*z[k]+y[j]*y[j])/(p.ar*p.ar)-x[i]*x[i]/(p.az*p.az))*p.phi1/(p.az*p.az)*y[j];
                }

                // Field for which we know the relaxed state analytically extending in Y.
                if (strncmp(p.bInit, "analyticY ", 10) == 0) {
                    // Field used for testing.
                    h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] =
                        2*p.ampl*y[j]*exp(-(x[i]*x[i]+z[k]*z[k])/(p.ar*p.ar)-y[j]*y[j]/(p.az*p.az))*p.phi1/(p.az*p.az)*z[k];
                    h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl;
                    h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] =
                        -2*p.ampl*y[j]*exp(-(x[i]*x[i]+z[k]*z[k])/(p.ar*p.ar)-y[j]*y[j]/(p.az*p.az))*p.phi1/(p.az*p.az)*x[i];
                }

                // Homogeneous magnetic field in z-direction.
                if (strncmp(p.bInit, "homZ ", 5) == 0) {
                    // homogeneous magnetic field
                    h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0.;
                    h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0.;
                    h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl;
                }

                // Blob configuration, like e1, e2 and e3 from Wilmot-Smith ApJ, 696:1339 (2009).
                if (strncmp(p.bInit, "blobs ", 6) == 0) {
                    h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0.;
                    h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0.;
                    h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0.;
                    for (b = 0; b < p.nBlobs; b++) {
                        h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] += 2*p.ampl*p.blobTwist[b]/p.blobScale[b]*
                                exp((-(x[i]-p.blobXc[b])*(x[i]-p.blobXc[b])-(y[j]-p.blobYc[b])*(y[j]-p.blobYc[b]))/(p.blobScale[b]*p.blobScale[b]) -
                                (z[k]-p.blobZc[b])*(z[k]-p.blobZc[b])/(p.blobZl[b]*p.blobZl[b])) *
                                (-(y[j]-p.blobYc[b]));
                        h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] += 2*p.ampl*p.blobTwist[b]/p.blobScale[b]*
                                exp((-(x[i]-p.blobXc[b])*(x[i]-p.blobXc[b])-(y[j]-p.blobYc[b])*(y[j]-p.blobYc[b]))/(p.blobScale[b]*p.blobScale[b]) -
                                (z[k]-p.blobZc[b])*(z[k]-p.blobZc[b])/(p.blobZl[b]*p.blobZl[b])) *
                                (x[i]-p.blobXc[b]);
                    }
                    h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl;
                }

                // Field for the shearing experiments.
                if (strncmp(p.bInit, "sheared ", 8) == 0) {
                    h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0.;
                    h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl*sin(p.initShearK*2*PI*(x[i]-p.Ox)/p.Lx)*0.5;
                    h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl*(1+0.25*pow(cos(p.initShearK*2*PI*(x[i]-p.Ox)/p.Lx),2));
                }

                // Field for null fan configuration from Phys. of Plasm. 12, 072112 (2005).
                if (strncmp(p.bInit, "nullFan ", 8) == 0) {
                    h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl*x[i]/2;
                    h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl*(y[j]/2-p.pert*z[k]);
                    h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = -p.ampl*z[k];
                }

                // Field for from ApJ 756:7 (6pp), 2012.
                if (strncmp(p.bInit, "fanSeparatrix ", 14) == 0) {
                    tmp = -0.6/sqrt(pow(x[i]*x[i] + pow(y[j]-0.02,2) + pow(z[k]+1.4,2),3));
                    h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl*x[i]*tmp;
                    h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl*(y[j]-0.02)*tmp;
                    h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl*(1+tmp*(z[k]+1.4));
                }

                // Twisted tube.
                if (strncmp(p.bInit, "twisted ", 8) == 0) {
//                    tmp = exp(-(pow(x[i]-p.pert*exp(-pow(z[k],2)/(p.az*p.az)),2)+pow(y[j],2))/(p.ar*p.ar));
                    tmp = exp(-(pow(x[i],2)+pow(y[j],2))/(p.ar*p.ar));
                    h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl*(tmp*y[j]*p.twist - 2*p.pert*z[k]*exp(-z[k]*z[k]/(p.az*p.az))/(p.az*p.az));
                    h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = -p.ampl*tmp*x[i]*p.twist;
                    h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl;
                }

                // Twisted tubes suggested by G. Hornig 2015.
                if (strncmp(p.bInit, "tubeSetA ", 9) == 0) {
                    r = sqrt(pow(x[i], 2) + pow(y[j], 2));
                    if (r > p.ar) {
                        h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0;
                        h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0;
                        h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0;
                    }
                    else {
                        if (r > 0) {
                            tmp = 4*pow(r/p.ar,3)/p.ar;    // B_p
                            h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl*tmp*y[j]/r;
                            h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = -p.ampl*tmp*x[i]/r;
                            h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl*(sin(PI*r/(2*p.ar)) + r*PI/(2*p.ar)*cos(PI*r/2/p.ar))/r;
                        }
                        else {
                            h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0;
                            h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0;
                            h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl*PI/p.ar;
                        }
                    }
                }
                if (strncmp(p.bInit, "tubeSetB ", 9) == 0) {
                    r = sqrt(pow(x[i], 2) + pow(y[j], 2));
                    if (r > p.ar) {
                        h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0;
                        h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0;
                        h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl/p.ar;
                    }
                    else {
                        if (r > 0) {
                            tmp = 4*pow(1-pow(r/p.ar,2),2)*r/(p.ar*p.ar);    // B_p
                            h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl*tmp*y[j]/r;
                            h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = -p.ampl*tmp*x[i]/r;
                            h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] =
                                    p.ampl*(sin(PI/2*pow(r/p.ar,2)) + pow(r/p.ar,2)*PI*cos(PI/2*pow(r/p.ar,2)))/r;
                        }
                        else {
                            h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0;
                            h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0;
                            h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0;
                        }
                    }
                }

                // Hopf field as described in Smiet (2015) 10.1103/PhysRevLett.115.095001.
                if (strncmp(p.bInit, "hopf ", 5) == 0) {
                    tmp = 4*p.ampl*pow(p.ar,4)/(PI*pow((p.ar*p.ar+x[i]*x[i]+y[j]*y[j]+z[k]*z[k]),3));
                    h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = tmp*2*(p.phi2*p.ar*y[j]-p.phi1*x[i]*z[k]);
                    h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = -tmp*2*(p.phi2*p.ar*x[i]+p.phi1*y[j]*z[k]);
                    h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = tmp*p.phi1*(-p.ar*p.ar+x[i]*x[i]+y[j]*y[j]-z[k]*z[k]);
                    h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] += p.bGround;
                }

        // Discontinuous field in z-direction with kink into x.
        if (strncmp(p.bInit, "discont ", 8) == 0) {
            if (z[k] > p.az) {
                h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.pert;
            }
            else {
                h.B0[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0;
            }
            h.B0[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = 0;
            h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = p.ampl;
        }

                // Set the initial grid to undistorted.
                h.xb[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = x[i];
                h.xb[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = y[j];
                h.xb[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] = z[k];
            }
        }
    }

    // Borromean rings
    if (strncmp(p.bInit, "Borromean ", 10) == 0) {
        REAL dEllipseParam, dCircleParam, dCircleRadius;
        REAL ellipseParam, circleParam, circleRadius;
        REAL ellipsePos[3], circlePos[3], tangent[3], normal[3];
        REAL len;
        int *nCompute; // array containing the weighting factors for the field smoothing

        // Compute the step lengths such that no grid cell is left out.
        dEllipseParam = min(min(p.dx, p.dy), p.dz) / (p.major+p.width/2.) / p.stretch / 4.;
        dCircleParam = min(min(p.dx, p.dy), p.dz)/(p.width/2.) / p.stretch / 4.;
        dCircleRadius = min(min(p.dx, p.dy), p.dz) / 4.;

        // Initialize the magnetic field to 0.
        memset(h.B0, 0, sizeof(h.B0));

        nCompute = (int *)malloc(p.nx*p.ny*p.nz*sizeof(*(nCompute)));
        if (nCompute == NULL) { printf("error: could not allocate memory for nCompute\n"); return -1; }
        memset(nCompute, 0, sizeof(nCompute));

        for (b = 0; b < 3; b++) {
            ellipseParam = 0.;
            while(ellipseParam <= 2*PI) {
                if (b == 0) {
                    ellipsePos[0] = p.major*sin(ellipseParam);
                    ellipsePos[1] = p.minor*cos(ellipseParam);
                    ellipsePos[2] = 0;
                    tangent[0] = p.major*cos(ellipseParam);
                    tangent[1] = -p.minor*sin(ellipseParam);
                    tangent[2] = 0;
                }
                if (b == 1) {
                    ellipsePos[0] = p.minor*cos(ellipseParam);
                    ellipsePos[1] = 0;
                    ellipsePos[2] = p.major*sin(ellipseParam);
                    tangent[0] = -p.minor*sin(ellipseParam);
                    tangent[1] = 0;
                    tangent[2] = p.major*cos(ellipseParam);
                }
                if (b == 2) {
                    ellipsePos[0] = 0;
                    ellipsePos[1] = p.major*sin(ellipseParam);
                    ellipsePos[2] = p.minor*cos(ellipseParam);
                    tangent[0] = 0;
                    tangent[1] = p.major*cos(ellipseParam);
                    tangent[2] = -p.minor*sin(ellipseParam);
                }
                len = sqrt(tangent[0]*tangent[0]+tangent[1]*tangent[1]+tangent[2]*tangent[2]);
                for (l = 0; l < 3; l++)
                    tangent[l] = tangent[l] / len;

                //  Find vector that is orthonormal to tangent vector.
                if (abs(tangent[0]) <= 0.5) {
                    normal[0] = 0;
                    normal[1] = tangent[2];
                    normal[2] = -tangent[1];
                }
                else if (abs(tangent[1]) <= 0.5) {
                    normal[0] = -tangent[2];
                    normal[1] = 0;
                    normal[2]= tangent[0];
                }
                else {
                    normal[0] = tangent[1];
                    normal[1] = -tangent[0];
                    normal[2] = 0;
                }

                // Normalize the normal vector.
                len = sqrt(normal[0]*normal[0]+normal[1]*normal[1]+normal[2]*normal[2]);
                for (l = 0; l < 3; l++)
                    normal[l] = normal[l] / len;

                circleRadius = 0.;

                // Loop which changes the circle's radius.
                while (circleRadius <= p.width/2.) {
                    circleParam = 0.;

                    // Loop which goes around the circle.
                    while (circleParam <= 2.*PI) {
                        circlePos[0] = ellipsePos[0] + circleRadius *
                                ((tangent[0]*tangent[0]*(1-cos(circleParam))+cos(circleParam))*normal[0] +
                                   (tangent[0]*tangent[1]*(1-cos(circleParam))-tangent[2]*sin(circleParam))*normal[1] +
                                (tangent[0]*tangent[2]*(1-cos(circleParam))+tangent[1]*sin(circleParam))*normal[2]);
                        circlePos[1] = ellipsePos[1] + circleRadius *
                                ((tangent[0]*tangent[1]*(1-cos(circleParam))+tangent[2]*sin(circleParam))*normal[0] +
                                (tangent[1]*tangent[1]*(1-cos(circleParam))+cos(circleParam))*normal[1] +
                                (tangent[1]*tangent[2]*(1-cos(circleParam))-tangent[0]*sin(circleParam))*normal[2]);
                        circlePos[2] = ellipsePos[2] + circleRadius *
                                ((tangent[0]*tangent[2]*(1-cos(circleParam))-tangent[1]*sin(circleParam))*normal[0] +
                                (tangent[1]*tangent[2]*(1-cos(circleParam))+tangent[0]*sin(circleParam))*normal[1] +
                                (tangent[2]*tangent[2]*(1-cos(circleParam))+cos(circleParam))*normal[2]);

                        // Find the corresponding mesh point to this position
                        i = int((circlePos[0]*p.stretch - p.Ox)*p.dx1) + 1;
                        j = int((circlePos[1]*p.stretch - p.Oy)*p.dy1) + 1;
                        k = int((circlePos[2]*p.stretch - p.Oz)*p.dz1) + 1;

                        if ((i >= 0) && (j >= 0) && (k >= 0) && (i < (p.nx+2)) && (j < (p.ny+2)) && (k < (p.nz+2)))
                            for (l = 0; l < 3; l++) {
                                h.B0[l + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] += tangent[l]*p.ampl*
                                        (exp(-(2*circleRadius/p.width)*(2*circleRadius/p.width))-exp(-1.)) / (1-exp(-1.));
                                nCompute[i + j*(p.nx+2) + k*(p.nx+2)*(p.ny+2)] += 1;
                            }

                        circleParam = circleParam + dCircleParam;
                    }
                    circleRadius = circleRadius + dCircleRadius;
                }
                ellipseParam = ellipseParam + dEllipseParam;
            }
        }

        // Add homogeneous magnetic field and do the averaging.
        for (k = 0; k < p.nz+2; k++)
            for (j = 0; j < p.ny+2; j++)
                for (i = 0; i < p.nx+2; i++) {
                    if (nCompute[i + j*(p.nx+2) + k*(p.nx+2)*(p.ny+2)] > 1)
                        for (l = 0; l < 3; l++)
                            h.B0[l + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] /= nCompute[i + j*(p.nx+2) + k*(p.nx+2)*(p.ny+2)];
                    h.B0[2 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] += p.bGround;
                }
    }


    //
    // velocity field uu
    //

    // Exclude the boundaries.
    if (p.inertia == 1) {
        if (strncmp(p.uInit, "nil ", 4) == 0)
            memset(h.B0, 0, sizeof(h.B0));

        // Adapt for own initial velocity field.
//        for (k = 0; k < p.nz; k++) {
//            z[k] = p.dz*k + p.Oz;
//            for (j = 0; j < p.ny; j++) {
//                y[j] = p.dy*j + p.Oy;
//                for (i = 0; i < p.nx; i++) {
//                    x[i] = p.dx*i + p.Ox;
//                    if (strncmp(p.uInit, "nil ", 4) == 0) {
//                        h.uu[0 + i*3 + j*p.nx*3 + k*p.nx*p.ny*3] = 0;
//                        h.uu[1 + i*3 + j*p.nx*3 + k*p.nx*p.ny*3] = 0;
//                        h.uu[2 + i*3 + j*p.nx*3 + k*p.nx*p.ny*3] = 0;
//                    }
//                }
//            }
//        }
    }

    return 0;
}


// Add a distortion to the initial grid xb. Note that B0 refers to the undistorted grid.
int initDistortion(REAL *xb, struct Parameters p)
{
    int  i, j, k;
    REAL x[p.nx+2], y[p.ny+2], z[p.nz+2], yy;

    if (strncmp(p.initDist, "none ", 5) != 0) {
        // Include the boundaries.
        for (k = 0; k < p.nz+2; k++) {
            z[k] = p.dz*(k-1) + p.Oz;
            for (j = 0; j < p.ny+2; j++) {
                y[j] = p.dy*(j-1) + p.Oy;
                for (i = 0; i < p.nx+2; i++) {
                    x[i] = p.dx*(i-1) + p.Ox;

                    // Sinusoidal shear.
                    if (strncmp(p.initDist, "initShearX ", 11) == 0) {
                        xb[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] -= p.initShearA * sin(p.initShearK*2*PI*(x[i]+p.Ox-p.dx/2)/(p.Lx+p.dx)) * z[k];
                        yy = xb[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3];
                        xb[0 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] -= p.initShearB * sin(p.initShearK*2*PI*(yy+p.Oy-p.dy/2)/(p.Ly+p.dy)) * z[k];
                    }

                    // Sinusoidal shear, works best for boxes of -1 < xyz < 1.
                    if (strncmp(p.initDist, "centerShift ", 12) == 0) {
                        xb[1 + i*3 + j*(p.nx+2)*3 + k*(p.nx+2)*(p.ny+2)*3] +=
                            p.initShearA * exp(-p.initShearK*x[i]*x[i])*(1-y[j]*y[j])*(exp(-1*z[k]+1) - 5.0804508111226507);
                    }
                }
            }
        }
    }
    return 0;
}

